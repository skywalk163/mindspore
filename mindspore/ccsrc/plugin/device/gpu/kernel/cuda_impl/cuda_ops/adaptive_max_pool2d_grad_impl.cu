#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/adaptive_max_pool2d_grad_impl.cuh"
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void AdaptiveMaxPool2DGradKernel(const T *input_data, const S *max_index, const int input_nchw,
                                            const int input_hw, const int output_hw, T *output_data) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < input_nchw; pos += blockDim.x * gridDim.x) {
    const S idx = max_index[pos];
    const int posn = pos / input_hw;
    MsAtomicAdd(output_data + posn * output_hw + static_cast<int>(idx), input_data[pos]);
  }
  return;
}

template <typename T, typename S>
hipError_t CalAdaptiveMaxPool2DGrad(const T *input_data, const S *max_index, const int n, const int c,
                                     const uint input_height, const uint input_width, const uint output_height,
                                     const uint output_width, T *output_data, const uint32_t &device_id,
                                     hipStream_t hip_stream) {
  const int input_hw = input_height * input_width;
  const int input_chw = c * input_hw;
  const int input_nchw = n * input_chw;
  const int output_hw = output_height * output_width;

  AdaptiveMaxPool2DGradKernel<<<CUDA_BLOCKS(device_id, input_nchw), CUDA_THREADS(device_id), 0, hip_stream>>>(
    input_data, max_index, input_nchw, input_hw, output_hw, output_data);
  return GetCudaStatus();
}

#define REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(type1, type2)                                                 \
  template CUDA_LIB_EXPORT hipError_t CalAdaptiveMaxPool2DGrad<type1, type2>(                          \
    const type1 *input_data, const type2 *max_index, const int n, const int c, const uint input_height, \
    const uint input_width, const uint output_height, const uint output_width, type1 *output_data,      \
    const uint32_t &device_id, hipStream_t hip_stream)

REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(half, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(float, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(double, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int8_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int16_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int32_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int64_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint8_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint16_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint32_t, int32_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint64_t, int32_t);

REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(half, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(float, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(double, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int8_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int16_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int32_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(int64_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint8_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint16_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint32_t, int64_t);
REG_ADAPTIVE_MAX_POOL2D_GRAD_CUDA(uint64_t, int64_t);

#include "hip/hip_runtime.h"
/**
 * Copyright 2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "maxpool_grad_with_argmax_v2_impl.cuh"
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void MaxPoolGradWithArgmaxV2(const T *dy, const S *index, const int64_t x_hw, const int64_t x_chw,
                                        const int64_t dy_hw, const int64_t dy_chw, const int64_t size, T *dx) {
  for (int64_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    const S idx = index[pos];
    const int64_t pos_n = pos / dy_chw;
    const int64_t pos_c = pos / dy_hw % (dy_chw / dy_hw);
    MsAtomicAdd(dx + pos_n * x_chw + pos_c * x_hw + idx, dy[pos]);
  }
  return;
}

template <typename T>
__global__ void InitOutput(const int size, T *output) {
  T zero = 0;
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < size; id += blockDim.x * gridDim.x) {
    output[id] = zero;
  }
  return;
}

template <typename T, typename S>
hipError_t CalMaxPoolGradWithArgmaxV2(const T *dy, const S *index, const int64_t x_hw, const int64_t x_chw,
                                       const int64_t x_nchw, const int64_t dy_hw, const int64_t dy_chw,
                                       const int64_t dy_nchw, T *dx, const uint32_t device_id,
                                       hipStream_t hip_stream) {
  InitOutput<<<GET_BLOCKS(x_nchw), GET_THREADS, 0, hip_stream>>>(x_nchw, dx);
  MaxPoolGradWithArgmaxV2<<<GET_BLOCKS(dy_nchw), GET_THREADS, 0, hip_stream>>>(dy, index, x_hw, x_chw, dy_hw, dy_chw,
                                                                                dy_nchw, dx);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<half, int32_t>(
  const half *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, half *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<float, int32_t>(
  const float *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, float *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<double, int32_t>(
  const double *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, double *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int8_t, int32_t>(
  const int8_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int8_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int16_t, int32_t>(
  const int16_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int16_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int32_t, int32_t>(
  const int32_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int32_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int64_t, int32_t>(
  const int64_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int64_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint8_t, int32_t>(
  const uint8_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint8_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint16_t, int32_t>(
  const uint16_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint16_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint32_t, int32_t>(
  const uint32_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint32_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint64_t, int32_t>(
  const uint64_t *dy, const int32_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint64_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<half, int64_t>(
  const half *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, half *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<float, int64_t>(
  const float *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, float *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<double, int64_t>(
  const double *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, double *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int8_t, int64_t>(
  const int8_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int8_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int16_t, int64_t>(
  const int16_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int16_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int32_t, int64_t>(
  const int32_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int32_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<int64_t, int64_t>(
  const int64_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, int64_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint8_t, int64_t>(
  const uint8_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint8_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint16_t, int64_t>(
  const uint16_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint16_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint32_t, int64_t>(
  const uint32_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint32_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradWithArgmaxV2<uint64_t, int64_t>(
  const uint64_t *dy, const int64_t *index, const int64_t x_hw, const int64_t x_chw, const int64_t x_nchw,
  const int64_t dy_hw, const int64_t dy_chw, const int64_t dy_nchw, uint64_t *dx, const uint32_t device_id,
  hipStream_t hip_stream);

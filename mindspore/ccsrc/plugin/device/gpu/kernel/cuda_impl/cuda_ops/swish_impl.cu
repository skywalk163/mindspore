#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/swish_impl.cuh"
#include <math.h>
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SwishKernel(size_t size, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output[pos] = input[pos] / (static_cast<T>(1) + exp(-input[pos]));
  }
}

template <typename T>
hipError_t Swish(size_t size, const T *input, T *output, hipStream_t hip_stream, const uint32_t device_id) {
  SwishKernel<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Swish<float>(size_t size, const float *input, float *output,
                                                  hipStream_t hip_stream, const uint32_t device_id);

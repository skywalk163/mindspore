#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2024 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/gatherd.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "include/hip/hip_fp16.h"

template <typename T>
using Complex = mindspore::utils::Complex<T>;

template <typename T, typename S>
__global__ void GatherDKernel(const T *input, const S *index, T *output, const ShapeHelper input_shape,
                              const ShapeHelper index_shape, size_t dim, size_t num, size_t rank) {
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < num; id += blockDim.x * gridDim.x) {
    S j = index[id];
    if (j < 0) {
      j += static_cast<S>(input_shape.shape[dim]);
    }
    CUDA_KERNEL_ASSERT(j >= 0);
    size_t j_read = static_cast<size_t>(j);
    CUDA_KERNEL_ASSERT(j_read < input_shape.shape[dim]);
    size_t offset = 0;
    size_t moved_id = id;
    size_t moved_offset = 1;
    for (size_t i = rank; i > 0; i--) {
      auto real_i = i - 1;
      auto cur_idx = moved_id % index_shape.shape[real_i];
      moved_id = moved_id / index_shape.shape[real_i];
      auto cur_input_idx = real_i == dim ? j_read : cur_idx;
      offset += cur_input_idx * moved_offset;
      moved_offset *= input_shape.shape[real_i];
    }
    output[id] = input[offset];
  }
  return;
}

template <typename T, typename S>
hipError_t GatherD(const T *input, const S *index, T *output, size_t dim, size_t num, size_t rank,
                    const ShapeHelper &input_shape, const ShapeHelper &index_shape, hipStream_t stream,
                    uint32_t device_id) {
  GatherDKernel<<<CUDA_BLOCKS(device_id, num), CUDA_THREADS(device_id), 0, stream>>>(input, index, output, input_shape,
                                                                                     index_shape, dim, num, rank);
  return GetCudaStatus();
}

#define SPECIALIZE_KERNEL(T, S)                                                                                     \
  template CUDA_LIB_EXPORT hipError_t GatherD<T, S>(                                                               \
    const T *input, const S *index, T *output, size_t dim, size_t num, size_t rank, const ShapeHelper &input_shape, \
    const ShapeHelper &index_shape, hipStream_t stream, uint32_t device_id);

SPECIALIZE_KERNEL(float, int64_t)
SPECIALIZE_KERNEL(Complex<double>, int)
SPECIALIZE_KERNEL(Complex<double>, int64_t)
SPECIALIZE_KERNEL(Complex<float>, int)
SPECIALIZE_KERNEL(Complex<float>, int64_t)
SPECIALIZE_KERNEL(double, int)
SPECIALIZE_KERNEL(double, int64_t)
SPECIALIZE_KERNEL(float, int)
SPECIALIZE_KERNEL(half, int)
SPECIALIZE_KERNEL(half, int64_t)
SPECIALIZE_KERNEL(int64_t, int)
SPECIALIZE_KERNEL(int64_t, int64_t)
SPECIALIZE_KERNEL(int, int)
SPECIALIZE_KERNEL(int, int64_t)
SPECIALIZE_KERNEL(int16_t, int)
SPECIALIZE_KERNEL(int16_t, int64_t)
SPECIALIZE_KERNEL(int8_t, int)
SPECIALIZE_KERNEL(int8_t, int64_t)
SPECIALIZE_KERNEL(unsigned char, int)
SPECIALIZE_KERNEL(unsigned char, int64_t)
SPECIALIZE_KERNEL(bool, int)
SPECIALIZE_KERNEL(bool, int64_t)
SPECIALIZE_KERNEL(uint16_t, int)
SPECIALIZE_KERNEL(uint16_t, int64_t)
SPECIALIZE_KERNEL(uint32_t, int)
SPECIALIZE_KERNEL(uint32_t, int64_t)
SPECIALIZE_KERNEL(uint64_t, int)
SPECIALIZE_KERNEL(uint64_t, int64_t)

#undef SPECIALIZE_KERNEL

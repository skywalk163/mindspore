#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/layer_norm_grad_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/layer_norm_impl.cuh"
#include "include/hip/hip_fp16.h"

constexpr int NUM_PER_THREAD_REDUCE = 4;
constexpr int WARP_SIZE = 32;

template <typename T>
inline __device__ T my_pow(T a, double b) {
  return pow(a, static_cast<float>(b));
}

template <>
inline __device__ half my_pow(half a, double b) {
  return __float2half(pow(__half2float(a), static_cast<float>(b)));
}

template <typename T>
inline __device__ void GammaAndBetaThreadReduce(const int col, const int row_dim, const int col_dim, const int mean_dim,
                                                const float epsilon, const T *dy, const T *x, const float *mean,
                                                const float *var, float *dg, float *db) {
  int loop_num = (row_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int row = NUM_PER_THREAD_REDUCE * i + j;
      if (row >= row_dim) {
        return;
      }

      int pos = row * col_dim + col;
      int mean_offset = pos / mean_dim;
      dg[0] += static_cast<float>(dy[pos]) * my_pow(var[mean_offset] + epsilon, -0.5) *
               (static_cast<float>(x[pos]) - mean[mean_offset]);
      db[0] += static_cast<float>(dy[pos]);
    }
  }
}

template <typename T>
inline __device__ void GammaAndBetaWarpReduce(T *dg, T *db) {
  for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
    dg[0] += __shfl_down_sync(0xffffffff, dg[0], delta);
    db[0] += __shfl_down_sync(0xffffffff, db[0], delta);
  }
}

template <typename T>
inline __device__ void GammaAndBetaBlockReduce(const int col, const int row_dim, float *dg, float *db, T *dg_addr,
                                               T *db_addr) {
  // load data to share memory
  // thread(0, 32, 64, 96, ...) keep the data
  DynamicSharedMem<float> share_mem;
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 2;
    share_mem.addr()[offset] = dg[0];
    share_mem.addr()[offset + 1] = db[0];
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 2;
      share_mem.addr()[threadIdx.x * 2] += share_mem.addr()[offset];
      share_mem.addr()[threadIdx.x * 2 + 1] += share_mem.addr()[offset + 1];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    dg_addr[col] = (T)(share_mem.addr()[0]);
    db_addr[col] = (T)(share_mem.addr()[1]);
  }
}

template <typename T>
__global__ void GammaAndBetaPropKernel(const int row_dim, const int col_dim, const int mean_dim, const float epsilon,
                                       const T *dy, const T *x, const float *mean_addr, const float *var_addr,
                                       T *dg_addr, T *db_addr) {
  // row: [0:param_axis]
  // col: [param_axis:]
  // dg[i][j] = dy[i][j] * (var[i] + epsilon, -0.5) * (x[i][j] - mean[i])
  // dg[j] = \Sigma_{j}dg[i][j]
  for (int col = blockIdx.x; col < col_dim; col += gridDim.x) {
    float dg = 0;
    float db = 0;
    GammaAndBetaThreadReduce(col, row_dim, col_dim, mean_dim, epsilon, dy, x, mean_addr, var_addr, &dg, &db);
    GammaAndBetaWarpReduce(&dg, &db);
    GammaAndBetaBlockReduce(col, row_dim, &dg, &db, dg_addr, db_addr);
  }
}

constexpr int kTileSize = 8;
template <typename T>
struct alignas(sizeof(T) * kTileSize) TArray {
  T data[kTileSize];
};

template <typename T>
inline __device__ void TiledGammaAndBetaThreadReduce(const int col, const int row_dim, const int col_dim,
                                                     const int mean_dim, const float epsilon, const T *dy, const T *x,
                                                     const float *mean, const float *var, float *dg, float *db) {
  for (int i = 0; i < kTileSize; ++i) {
    dg[i] = 0;
    db[i] = 0;
  }
  for (int i = threadIdx.x; i < row_dim; i += blockDim.x) {
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[i * col_dim + col]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[i * col_dim + col]);
    float var_rsqrt = my_pow(var[i] + epsilon, -0.5);
    for (int j = 0; j < kTileSize; ++j) {
      dg[j] += static_cast<float>(dy_tile[j]) * var_rsqrt * (static_cast<float>(x_tile[j]) - mean[i]);
      db[j] += static_cast<float>(dy_tile[j]);
    }
  }
}

template <typename T>
inline __device__ void TiledGammaAndBetaWarpReduce(T *dg, T *db) {
  for (int i = 0; i < kTileSize; ++i) {
    for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
      dg[i] += __shfl_down_sync(0xffffffff, dg[i], delta);
      db[i] += __shfl_down_sync(0xffffffff, db[i], delta);
    }
  }
}

template <typename T>
inline __device__ void TiledGammaAndBetaBlockReduce(const int col, const int row_dim, float *dg, float *db, T *dg_addr,
                                                    T *db_addr) {
  DynamicSharedMem<float> share_mem;
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 2 * kTileSize;
    for (int i = 0; i < kTileSize; ++i) {
      share_mem.addr()[offset + i * 2] = dg[i];
      share_mem.addr()[offset + i * 2 + 1] = db[i];
    }
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 2 * kTileSize;
      for (int i = 0; i < kTileSize; ++i) {
        share_mem.addr()[threadIdx.x * 2 * kTileSize + 2 * i] += share_mem.addr()[offset + 2 * i];
        share_mem.addr()[threadIdx.x * 2 * kTileSize + 2 * i + 1] += share_mem.addr()[offset + 2 * i + 1];
      }
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < kTileSize; ++i) {
      dg_addr[col + i] = (T)(share_mem.addr()[2 * i]);
      db_addr[col + i] = (T)(share_mem.addr()[2 * i + 1]);
    }
  }
}

template <typename T>
__global__ void TiledGammaAndBetaPropKernel(const int row_dim, const int col_dim, const int mean_dim,
                                            const float epsilon, const T *dy, const T *x, const float *mean_addr,
                                            const float *var_addr, T *dg_addr, T *db_addr) {
  for (int col = blockIdx.x * kTileSize; col < col_dim; col += gridDim.x * kTileSize) {
    float dg[kTileSize];
    float db[kTileSize];
    TiledGammaAndBetaThreadReduce(col, row_dim, col_dim, mean_dim, epsilon, dy, x, mean_addr, var_addr, dg, db);
    TiledGammaAndBetaWarpReduce(dg, db);
    TiledGammaAndBetaBlockReduce(col, row_dim, dg, db, dg_addr, db_addr);
  }
}

template <typename T>
inline __device__ void InputThreadReduce(const int row, const int col_dim, const int param_dim, const float epsilon,
                                         float *sum1, float *sum2, float *sum3, const T *dy, const T *x,
                                         const float *mean, const float *var, const T *gamma) {
  int loop_num = (col_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int col = NUM_PER_THREAD_REDUCE * i + j;
      if (col >= col_dim) {
        sum1[0] = -0.5 * sum1[0] * my_pow(var[row] + epsilon, -1.5);
        sum3[0] = -2.0 * sum3[0];
        return;
      }

      int pos = row * col_dim + col;
      int gamma_offset = pos % param_dim;
      float v1 = static_cast<float>(dy[pos] * gamma[gamma_offset]);
      float v2 = static_cast<float>(x[pos]) - mean[row];

      sum1[0] += v1 * v2;
      sum2[0] += v1;
      sum3[0] += v2;
    }
  }
  sum1[0] = -0.5 * sum1[0] * my_pow(var[row] + epsilon, -1.5);
  sum3[0] = -2.0 * sum3[0];
}

template <typename T>
inline __device__ void TiledInputThreadReduce(const int row, const int col_dim, const int param_dim,
                                              const float epsilon, float *sum1, float *sum2, float *sum3, const T *dy,
                                              const T *x, const float *mean, const float *var, const T *gamma) {
  for (int i = threadIdx.x * kTileSize; i < col_dim; i += blockDim.x * kTileSize) {
    int pos = row * col_dim + i;
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[pos]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[pos]);

    for (int j = 0; j < kTileSize; ++j) {
      float v1 = static_cast<float>(dy_tile[j] * gamma[i + j]);
      float v2 = static_cast<float>(x_tile[j]) - mean[row];
      sum1[0] += v1 * v2;
      sum2[0] += v1;
      sum3[0] += v2;
    }
  }
  sum1[0] = -0.5 * sum1[0] * my_pow(var[row] + epsilon, -1.5);
  sum3[0] = -2.0 * sum3[0];
}

template <typename T>
inline __device__ void InputWarpReduce(T *sum1, T *sum2, T *sum3) {
  for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
    sum1[0] += __shfl_down_sync(0xffffffff, sum1[0], delta);
    sum2[0] += __shfl_down_sync(0xffffffff, sum2[0], delta);
    sum3[0] += __shfl_down_sync(0xffffffff, sum3[0], delta);
  }
}

template <typename T>
inline __device__ void InputBlockReduce(const int col_dim, T *sum1, T *sum2, T *sum3, T *share_mem) {
  // load data to share memory
  // thread(0, 32, 64, 96, ...) keep the data
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 3;
    share_mem[offset] = sum1[0];
    share_mem[offset + 1] = sum2[0];
    share_mem[offset + 2] = sum3[0];
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 3;
      share_mem[threadIdx.x * 3] += share_mem[offset];
      share_mem[threadIdx.x * 3 + 1] += share_mem[offset + 1];
      share_mem[threadIdx.x * 3 + 2] += share_mem[offset + 2];
    }
  }
  __syncthreads();
}

template <typename T>
inline __device__ void InputProp(const int row, const int col_dim, const int param_dim, const float epsilon,
                                 const T *dy, const T *x, const float *mean, const float *var, const T *gamma, T *dx,
                                 const float *share_mem) {
  float v3 = my_pow(var[row] + epsilon, -0.5);
  float v4 = share_mem[0] * (2.0 / col_dim);
  float v5 = (-1.0 * v3 * share_mem[1] + (1.0 / col_dim) * share_mem[0] * share_mem[2]) * (1.0 / col_dim);
  for (int col = threadIdx.x; col < col_dim; col += blockDim.x) {
    int pos = (row * col_dim + col);
    int gamma_offset = pos % param_dim;
    float v1 = static_cast<float>(dy[pos] * gamma[gamma_offset]);
    float v2 = static_cast<float>(x[pos]) - mean[row];
    dx[pos] = (T)(v1 * v3 + v4 * v2 + v5);
  }
}

template <typename T>
__global__ void InputPropKernel(const int row_dim, const int col_dim, const int param_dim, const float epsilon,
                                const T *dy, const T *x, const float *mean, const float *var, const T *gamma, T *dx) {
  for (int row = blockIdx.x; row < row_dim; row += gridDim.x) {
    float sum1 = 0;
    float sum2 = 0;
    float sum3 = 0;
    DynamicSharedMem<float> share_mem;
    InputThreadReduce(row, col_dim, param_dim, epsilon, &sum1, &sum2, &sum3, dy, x, mean, var, gamma);
    InputWarpReduce(&sum1, &sum2, &sum3);
    InputBlockReduce(col_dim, &sum1, &sum2, &sum3, share_mem.addr());
    InputProp(row, col_dim, param_dim, epsilon, dy, x, mean, var, gamma, dx, share_mem.addr());
  }
}

template <typename T>
inline __device__ void TiledInputProp(const int row, const int col_dim, const int param_dim, const float epsilon,
                                      const T *dy, const T *x, const float *mean, const float *var, const T *gamma,
                                      T *dx, const float *share_mem) {
  float col_inv = 1.0f / col_dim;
  float v3 = my_pow(var[row] + epsilon, -0.5);
  float v4 = share_mem[0] * col_inv * 2;
  float v5 = (col_inv * share_mem[0] * share_mem[2] - v3 * share_mem[1]) * col_inv;
  for (int col = threadIdx.x * kTileSize; col < col_dim; col += blockDim.x * kTileSize) {
    int pos = row * col_dim + col;
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    T dx_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[pos]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[pos]);

    for (int j = 0; j < kTileSize; ++j) {
      float v1 = static_cast<float>(dy_tile[j] * gamma[col + j]);
      float v2 = static_cast<float>(x_tile[j]) - mean[row];
      dx_tile[j] = (T)(v1 * v3 + v4 * v2 + v5);
    }
    TArray<T> *dx_tmp = reinterpret_cast<TArray<T> *>(&dx[pos]);
    *dx_tmp = *reinterpret_cast<TArray<T> *>(dx_tile);
  }
}

template <typename T>
__global__ void TiledInputPropKernel(const int row_dim, const int col_dim, const int param_dim, const float epsilon,
                                     const T *dy, const T *x, const float *mean, const float *var, const T *gamma,
                                     T *dx) {
  for (int row = blockIdx.x; row < row_dim; row += gridDim.x) {
    float sum1 = 0;
    float sum2 = 0;
    float sum3 = 0;
    TiledInputThreadReduce(row, col_dim, param_dim, epsilon, &sum1, &sum2, &sum3, dy, x, mean, var, gamma);
    InputWarpReduce(&sum1, &sum2, &sum3);
    DynamicSharedMem<float> share_mem;
    InputBlockReduce(col_dim, &sum1, &sum2, &sum3, share_mem.addr());
    TiledInputProp(row, col_dim, param_dim, epsilon, dy, x, mean, var, gamma, dx, share_mem.addr());
  }
}

template <typename T>
hipError_t LayerNormGrad(const int row_dim, const int col_dim, const int param_dim, const float epsilon, const T *dy,
                          const T *x, const float *mean, const float *var, const T *gamma, T *dx, T *dg, T *db,
                          hipStream_t stream) {
  const int thread_per_block = 256;
  int share_mem_size = thread_per_block / WARP_SIZE * 3 * sizeof(float);

  int param_reduce_dim = row_dim * col_dim / param_dim;
  int grid_size = param_dim;
  if (col_dim == param_dim && grid_size % kTileSize == 0 && col_dim % kTileSize == 0) {
    TiledInputPropKernel<<<row_dim, thread_per_block, share_mem_size, stream>>>(row_dim, col_dim, param_dim, epsilon,
                                                                                dy, x, mean, var, gamma, dx);
    share_mem_size = thread_per_block / WARP_SIZE * 2 * kTileSize * sizeof(float);
    grid_size /= kTileSize;
    TiledGammaAndBetaPropKernel<<<grid_size, thread_per_block, share_mem_size, stream>>>(
      param_reduce_dim, param_dim, col_dim, epsilon, dy, x, mean, var, dg, db);
  } else {
    InputPropKernel<<<row_dim, thread_per_block, share_mem_size, stream>>>(row_dim, col_dim, param_dim, epsilon, dy, x,
                                                                           mean, var, gamma, dx);
    share_mem_size = thread_per_block / WARP_SIZE * 2 * sizeof(float);
    GammaAndBetaPropKernel<<<grid_size, thread_per_block, share_mem_size, stream>>>(
      param_reduce_dim, param_dim, col_dim, epsilon, dy, x, mean, var, dg, db);
  }
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t LayerNormGrad(const int row_dim, const int col_dim, const int param_dim,
                                                   const float epsilon, const float *dy, const float *x,
                                                   const float *mean, const float *var, const float *gamma, float *dx,
                                                   float *dg, float *db, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t LayerNormGrad(const int row_dim, const int col_dim, const int param_dim,
                                                   const float epsilon, const half *dy, const half *x,
                                                   const float *mean, const float *var, const half *gamma, half *dx,
                                                   half *dg, half *db, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t LayerNormGrad(const int row_dim, const int col_dim, const int param_dim,
                                                   const float epsilon, const double *dy, const double *x,
                                                   const float *mean, const float *var, const double *gamma, double *dx,
                                                   double *dg, double *db, hipStream_t stream);

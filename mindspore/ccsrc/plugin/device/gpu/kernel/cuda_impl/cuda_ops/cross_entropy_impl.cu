#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "cross_entropy_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void CrossEntropyWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                             const size_t class_num, T *loss) {
  double total_loss = 0.0;
  T epsilon = 1e-6;
  for (size_t i = 0; i < batch_size; ++i) {
    T logit = logits[i * class_num + labels[i]];
    if (logit <= 0) {
      logit = epsilon;
    }
    total_loss += -logf(logit);
  }
  total_loss /= batch_size;
  loss[0] = static_cast<T>(total_loss);
}

template <typename T, typename S>
__global__ void LargeBatchCrossEntropyWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                       const size_t class_num, T *loss) {
  *loss = 0;
  T epsilon = 1e-6;
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size; index += blockDim.x * gridDim.x) {
    T logit = logits[index * class_num + labels[index]];
    if (logit <= 0) {
      logit = epsilon;
    }
    MsAtomicAdd(loss, -logf(logit) / batch_size);
  }
}

template <typename T, typename S>
__global__ void CrossEntropyGradWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                 const size_t class_num, T *grad) {
  for (size_t i = 0; i < class_num; i++) {
    for (size_t j = blockIdx.x * blockDim.x + threadIdx.x; j < batch_size; j += blockDim.x * gridDim.x) {
      if (labels[j] == i) {
        grad[j * class_num + i] = (logits[j * class_num + i] - 1) / batch_size;
      } else {
        grad[j * class_num + i] = logits[j * class_num + i] / batch_size;
      }
    }
  }
}

template <typename T, typename S>
__global__ void CrossEntropyKernel(const T *input0, const S *input1, const size_t batch_size, const size_t class_num,
                                   T *output0, T *output1, T *work) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size; index += blockDim.x * gridDim.x) {
    const float *logits = input0 + index * class_num;
    const float *labels = input1 + index * class_num;
    float *backprop = output1 + index * class_num;
    float *workspace = work + index * class_num;

    float maxv = logits[0];
    for (size_t i = 0; i < class_num; i++) {
      maxv = maxv > logits[i] ? maxv : logits[i];
    }

    float sum = 0.0;
    for (size_t i = 0; i < class_num; i++) {
      backprop[i] = logits[i] - maxv;
      workspace[i] = exp(backprop[i]);
      sum += workspace[i];
    }

    float logit = logf(sum);
    float loss = 0.0;

    for (size_t i = 0; i < class_num; i++) {
      loss += labels[i] * (backprop[i] - logit);
      workspace[i] = workspace[i] / sum;
      backprop[i] = workspace[i] - labels[i];
    }
    output0[index] = -loss;
  }
}

template <typename T, typename S>
hipError_t CrossEntropyWithSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                                   T *loss, hipStream_t hip_stream) {
  if (batch_size <= kLargeBatchLowLimit) {
    CrossEntropyWithSparseKernel<<<1, 1, 0, hip_stream>>>(logits, labels, batch_size, class_num, loss);
  } else {
    LargeBatchCrossEntropyWithSparseKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(
      logits, labels, batch_size, class_num, loss);
  }
  return GetCudaStatus();
}

template <typename T, typename S>
hipError_t CrossEntropyGradWithSparse(const T *logits, const S *labels, const size_t batch_size,
                                       const size_t class_num, T *grad, hipStream_t hip_stream) {
  CrossEntropyGradWithSparseKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(logits, labels, batch_size,
                                                                                            class_num, grad);
  return GetCudaStatus();
}

template <typename T, typename S>
hipError_t CrossEntropy(const T *logits, const S *labels, const size_t batch_size, const size_t class_num, T *losses,
                         T *dlogits, T *workspace, hipStream_t hip_stream) {
  CrossEntropyKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(logits, labels, batch_size, class_num,
                                                                              losses, dlogits, workspace);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CrossEntropyWithSparse<float, int>(const float *logits, const int *labels,
                                                                        const size_t batch_size, const size_t class_num,
                                                                        float *loss, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CrossEntropyWithSparse<float, int64_t>(const float *logits, const int64_t *labels,
                                                                            const size_t batch_size,
                                                                            const size_t class_num, float *loss,
                                                                            hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CrossEntropyGradWithSparse<float, int>(const float *logits, const int *labels,
                                                                            const size_t batch_size,
                                                                            const size_t class_num, float *grad,
                                                                            hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CrossEntropyGradWithSparse<float, int64_t>(const float *logits,
                                                                                const int64_t *labels,
                                                                                const size_t batch_size,
                                                                                const size_t class_num, float *grad,
                                                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CrossEntropy<float, float>(const float *logits, const float *labels,
                                                                const size_t batch_size, const size_t class_num,
                                                                float *losses, float *dlogits, float *workspace,
                                                                hipStream_t hip_stream);

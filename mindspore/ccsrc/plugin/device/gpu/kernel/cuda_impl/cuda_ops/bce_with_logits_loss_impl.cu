#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include <vector>
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/bce_with_logits_loss_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/elementwise/elementswise_pub_impl.cuh"

struct StrideInfo {
  size_t inp_stride[9];
  size_t out_stride[9];
  size_t index_stride[9];
};

template <typename T>
struct FillWithoutBroadcastFunctor {
  FillWithoutBroadcastFunctor() {}
  __device__ __forceinline__ void operator()(T *dst, const T *src) const { dst[0] = src[0]; }
};

StrideInfo CalBceStride(const std::vector<int64_t> inp_shape, const std::vector<int64_t> out_shape) {
  StrideInfo strides;
  size_t out_shape_size = out_shape.size();
  size_t cur_inp_shape = 0;
  if (out_shape_size == 0) {
    strides.inp_stride[0] = 1;
    strides.inp_stride[1] = 1;
    strides.out_stride[0] = 1;
    strides.out_stride[1] = 1;
    strides.index_stride[0] = 0;
    return strides;
  }
  strides.inp_stride[out_shape_size] = 1;
  strides.out_stride[out_shape_size] = 1;
  for (int idx = out_shape_size - 1; idx >= 0; --idx) {
    strides.inp_stride[idx] = strides.inp_stride[idx + 1] * inp_shape[idx];
    strides.out_stride[idx] = strides.out_stride[idx + 1] * out_shape[idx];
    cur_inp_shape = strides.inp_stride[idx] / strides.inp_stride[idx + 1];
    strides.index_stride[idx] = (cur_inp_shape == 1) ? 0 : 1;
  }
  return strides;
}

template <typename T>
__global__ void FillAndBroadcast(const size_t size, const size_t shape_size, const StrideInfo strides, const T *src,
                                 T *dst) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    size_t tmp_pos = pos;
    size_t cur_idx = 0;
    size_t input_pos = 0;
    for (int idx = 0; idx < shape_size; ++idx) {
      cur_idx = tmp_pos / strides.out_stride[idx + 1];
      input_pos += cur_idx * strides.inp_stride[idx + 1] * strides.index_stride[idx];
      tmp_pos -= cur_idx * strides.out_stride[idx + 1];
    }
    dst[pos] = src[input_pos];
  }
  return;
}

template <typename T>
struct BCEWithLogitsLossMainFunctor {
  BCEWithLogitsLossMainFunctor() {}
  __device__ __forceinline__ void operator()(T *output, const T *predict, const T *target,
                                             const T *shape_broadcasted) const {
    T max_value = -predict[0];
    max_value = max_value > static_cast<T>(0) ? max_value : static_cast<T>(0);
    const T log_weight = (shape_broadcasted[0] - static_cast<T>(1)) * target[0] + static_cast<T>(1);
    output[0] = (static_cast<T>(1) - target[0]) * predict[0] +
                log_weight * (log(exp(-max_value) + exp(-predict[0] - max_value)) + max_value);
  }
};

template <>
struct BCEWithLogitsLossMainFunctor<half> {
  BCEWithLogitsLossMainFunctor() {}
  __device__ __forceinline__ void operator()(half *output, const half *predict, const half *target,
                                             const half *shape_broadcasted) const {
    half max_value = -predict[0];
    max_value = max_value > static_cast<half>(0) ? max_value : static_cast<half>(0);
    const half log_weight = (shape_broadcasted[0] - static_cast<half>(1)) * target[0] + static_cast<half>(1);
    output[0] = (static_cast<half>(1) - target[0]) * predict[0] +
                log_weight * (hlog(hexp(-max_value) + hexp(-predict[0] - max_value)) + max_value);
  }
};

template <typename T>
struct MulFunctor {
  MulFunctor() {}
  __device__ __forceinline__ void operator()(T *rhs, const T *lhs) const { rhs[0] *= lhs[0]; }
};

template <typename T>
hipError_t CalBCEWithLogitsLoss(const size_t input_size, const T *predict, const T *target,
                                 const std::vector<int64_t> &input_shape, const size_t shape_size, const T *weight,
                                 const std::vector<int64_t> &weight_shape, const bool weight_need_broadcast,
                                 const T *pos_weight, const std::vector<int64_t> &pos_weight_shape,
                                 const bool pos_weight_need_broadcast, T *shape_broadcasted, T *output,
                                 hipStream_t hip_stream) {
  if (pos_weight_need_broadcast) {
    StrideInfo strides = CalBceStride(pos_weight_shape, input_shape);
    FillAndBroadcast<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(input_size, shape_size, strides,
                                                                              pos_weight, shape_broadcasted);
  } else {
    FillWithoutBroadcastFunctor<T> functor;
    cuda::elementwise::EltWiseCudaOpsFunc(functor, (uint)(input_size), shape_broadcasted, pos_weight, hip_stream);
  }
  BCEWithLogitsLossMainFunctor<T> loss_functor;
  cuda::elementwise::EltWiseCudaOpsFunc(loss_functor, (uint)(input_size), output, predict, target, shape_broadcasted,
                                        hip_stream);
  if (weight_need_broadcast) {
    StrideInfo strides = CalBceStride(weight_shape, input_shape);
    FillAndBroadcast<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(input_size, shape_size, strides, weight,
                                                                              shape_broadcasted);
  } else {
    FillWithoutBroadcastFunctor<T> functor;
    cuda::elementwise::EltWiseCudaOpsFunc(functor, (uint)(input_size), shape_broadcasted, weight, hip_stream);
  }
  MulFunctor<T> functor;
  cuda::elementwise::EltWiseCudaOpsFunc(functor, (uint)(input_size), output, shape_broadcasted, hip_stream);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalBCEWithLogitsLoss<half>(
  const size_t input_size, const half *predict, const half *target, const std::vector<int64_t> &input_shape,
  const size_t shape_size, const half *weight, const std::vector<int64_t> &weight_shape,
  const bool weight_need_broadcast, const half *pos_weight, const std::vector<int64_t> &pos_weight_shape,
  const bool pos_weight_need_broadcast, half *shape_broadcasted, half *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalBCEWithLogitsLoss<float>(
  const size_t input_size, const float *predict, const float *target, const std::vector<int64_t> &input_shape,
  const size_t shape_size, const float *weight, const std::vector<int64_t> &weight_shape,
  const bool weight_need_broadcast, const float *pos_weight, const std::vector<int64_t> &pos_weight_shape,
  const bool pos_weight_need_broadcast, float *shape_broadcasted, float *output, hipStream_t hip_stream);

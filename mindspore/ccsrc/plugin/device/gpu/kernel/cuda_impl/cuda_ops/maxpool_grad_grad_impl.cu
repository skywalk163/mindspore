#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/maxpool_grad_grad_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void MaxPoolGradGrad(const T *input, const T *grad, const int n, const int c, const int h, const int w,
                                const int windowHeight, const int windowWidth, const int strideHeight,
                                const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
                                const int outputWidth, const int outputNCHW, const int outputCHW, const int outputHW,
                                T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (outputNCHW); pos += blockDim.x * gridDim.x) {
    const int posn = pos / outputCHW;
    const int posc = pos / outputHW % c;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;
    int hstart = posh * strideHeight - padTop;
    int wstart = posw * strideWidth - padLeft;
    const int hend = min(hstart + windowHeight, h);
    const int wend = min(wstart + windowWidth, w);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    int inputStart = posn * c * h * w + posc * h * w;
    int maxIdx = hstart * w + wstart;
    T maxData = input[inputStart + maxIdx];
    for (int hcur = hstart; hcur < hend; ++hcur) {
      for (int wcur = wstart; wcur < wend; ++wcur) {
        int inputIdx = hcur * w + wcur;
        T inputData = input[inputStart + inputIdx];
        if (inputData > maxData) {
          maxIdx = inputIdx;
          maxData = inputData;
        }
      }
    }
    output[pos] = grad[inputStart + maxIdx];
  }
}

template <typename T>
hipError_t CalMaxPoolGradGrad(const T *input, const T *grad, const int n, const int c, const int h, const int w,
                               const int windowHeight, const int windowWidth, const int strideHeight,
                               const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
                               const int outputWidth, T *output, const uint32_t &device_id, hipStream_t hip_stream) {
  const int outputNCHW = n * c * outputHeight * outputWidth;
  const int outputCHW = c * outputHeight * outputWidth;
  const int outputHW = outputHeight * outputWidth;
  MaxPoolGradGrad<<<CUDA_BLOCKS(device_id, n * c * outputHeight * outputWidth), CUDA_THREADS(device_id), 0,
                    hip_stream>>>(input, grad, n, c, h, w, windowHeight, windowWidth, strideHeight, strideWidth,
                                   padTop, padLeft, outputHeight, outputWidth, outputNCHW, outputCHW, outputHW, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradGrad<float>(
  const float *input, const float *grad, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, float *output, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolGradGrad<half>(
  const half *input, const half *grad, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, half *output, const uint32_t &device_id, hipStream_t hip_stream);

template <typename T>
__global__ void MaxPool3DGradGrad(const T *input, const T *grad, const int n, const int c, const int d, const int h,
                                  const int w, const int windowDepth, const int windowHeight, const int windowWidth,
                                  const int strideDepth, const int strideHeight, const int strideWidth,
                                  const int padFront, const int padTop, const int padLeft, const int outputDepth,
                                  const int outputHeight, const int outputWidth, T *output) {
  const int outputNCDHW = n * c * outputDepth * outputHeight * outputWidth;
  const int outputCDHW = c * outputDepth * outputHeight * outputWidth;
  const int outputDHW = outputDepth * outputHeight * outputWidth;
  const int outputHW = outputHeight * outputWidth;

  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (outputNCDHW); pos += blockDim.x * gridDim.x) {
    const int posn = pos / outputCDHW;
    const int posc = pos / outputDHW % c;
    const int posd = pos / outputHW % outputDepth;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;

    int dstart = posd * strideDepth - padFront;
    int hstart = posh * strideHeight - padTop;
    int wstart = posw * strideWidth - padLeft;
    const int dend = min(dstart + windowDepth, d);
    const int hend = min(hstart + windowHeight, h);
    const int wend = min(wstart + windowWidth, w);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    int inputStart = posn * c * d * h * w + posc * d * h * w;
    int maxIdx = dstart * h * w + hstart * w + wstart;
    T maxData = input[inputStart + maxIdx];
    for (int dcur = dstart; dcur < dend; ++dcur) {
      for (int hcur = hstart; hcur < hend; ++hcur) {
        for (int wcur = wstart; wcur < wend; ++wcur) {
          int inputIdx = dcur * h * w + hcur * w + wcur;
          T inputData = input[inputStart + inputIdx];
          if (inputData > maxData) {
            maxIdx = inputIdx;
            maxData = inputData;
          }
        }
      }
    }
    output[pos] = grad[inputStart + maxIdx];
  }
}

template <typename T>
hipError_t CalMaxPool3DGradGrad(const T *input, const T *grad, const int n, const int c, const int d, const int h,
                                 const int w, const int windowDepth, const int windowHeight, const int windowWidth,
                                 const int strideDepth, const int strideHeight, const int strideWidth,
                                 const int padFront, const int padTop, const int padLeft, const int outputDepth,
                                 const int outputHeight, const int outputWidth, T *output, const uint32_t &device_id,
                                 hipStream_t hip_stream) {
  MaxPool3DGradGrad<<<CUDA_BLOCKS(device_id, n * c * outputDepth * outputHeight * outputWidth), CUDA_THREADS(device_id),
                      0, hip_stream>>>(input, grad, n, c, d, h, w, windowDepth, windowHeight, windowWidth, strideDepth,
                                        strideHeight, strideWidth, padFront, padTop, padLeft, outputDepth, outputHeight,
                                        outputWidth, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalMaxPool3DGradGrad<float>(
  const float *input, const float *grad, const int n, const int c, const int d, const int h, const int w,
  const int windowDepth, const int windowHeight, const int windowWidth, const int strideDepth, const int strideHeight,
  const int strideWidth, const int padFront, const int padTop, const int padLeft, const int outputDepth,
  const int outputHeight, const int outputWidth, float *output, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPool3DGradGrad<half>(
  const half *input, const half *grad, const int n, const int c, const int d, const int h, const int w,
  const int windowDepth, const int windowHeight, const int windowWidth, const int strideDepth, const int strideHeight,
  const int strideWidth, const int padFront, const int padTop, const int padLeft, const int outputDepth,
  const int outputHeight, const int outputWidth, half *output, const uint32_t &device_id, hipStream_t hip_stream);

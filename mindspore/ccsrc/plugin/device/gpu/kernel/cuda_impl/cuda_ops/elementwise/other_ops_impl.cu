#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHType WARRANTIES OR CONDITIONS OF ANY KTypeD, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <unsupported/Eigen/SpecialFunctions>
#include <limits>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/elementwise/eltwise_ops_func.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/elementwise/elt_unary_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/elementwise/elt_binary_impl.cuh"

template <typename Type>
struct UnaryFunc<ElwiseOpType::kErfinv, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return erfinv(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErfinv, float, float> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE float operator()(const float val) const { return erfinvf(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErfinv, half, half> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE half operator()(const half val) const { return __float2half(erfinvf(__half2float(val))); }
};
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kErfinv);

template <typename Type>
struct UnaryFunc<ElwiseOpType::kErf, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return erf(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErf, float, float> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE float operator()(const float val) const { return erff(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErf, half, half> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE half operator()(const half val) const { return __float2half(erff(__half2float(val))); }
};
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kErf);

template <typename Type>
struct UnaryFunc<ElwiseOpType::kErfc, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return erfc(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErfc, float, float> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE float operator()(const float val) const { return erfcf(val); }
};

template <>
struct UnaryFunc<ElwiseOpType::kErfc, half, half> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE half operator()(const half val) const { return __float2half(erfcf(__half2float(val))); }
};
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kErfc);

template <typename Type>
struct UnaryFunc<ElwiseOpType::kInvert, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return ~val; }
};
REGISTER_UNARY_OP_CUDA_FUNC_BOOL_TYPE(ElwiseOpType::kInvert);
REGISTER_UNARY_OP_CUDA_FUNC_INT_TYPE(ElwiseOpType::kInvert);
REGISTER_UNARY_OP_CUDA_FUNC_UINT_TYPE(ElwiseOpType::kInvert);
template <typename Type>
struct UnaryFunc<ElwiseOpType::kSign, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const {
    return val < Type(0.0) ? Type(-1.0) : (val > Type(0.0) ? Type(1.0) : Type(0.0));
  }
};

template <typename Type>
struct UnaryFunc<ElwiseOpType::kSign, Complex<Type>, Complex<Type>> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Complex<Type> operator()(const Complex<Type> val) const {
    Type sum = cuda::elwise::Sqrt<Type>(val.real() * val.real() + val.imag() * val.imag());
    if (sum != static_cast<Type>(0.0)) {
      return Complex<Type>(val.real() / sum, val.imag() / sum);
    } else {
      return Complex<Type>(0.0);
    }
  }
};
REGISTER_UNARY_OP_CUDA_FUNC_INT_TYPE(ElwiseOpType::kSign);
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kSign);
REGISTER_UNARY_OP_CUDA_FUNC_COMPLEX_TYPE(ElwiseOpType::kSign);
template <typename Type>
struct UnaryFunc<ElwiseOpType::kNeg, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return Type(-1) * val; }
};
REGISTER_UNARY_OP_CUDA_FUNC_BOOL_TYPE(ElwiseOpType::kNeg);
REGISTER_UNARY_OP_CUDA_FUNC_INT_TYPE(ElwiseOpType::kNeg);
REGISTER_UNARY_OP_CUDA_FUNC_UINT_TYPE(ElwiseOpType::kNeg);
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kNeg);
REGISTER_UNARY_OP_CUDA_FUNC_COMPLEX_TYPE(ElwiseOpType::kNeg);

template <typename Type>
struct UnaryFunc<ElwiseOpType::kReciprocal, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const {
    if (val != Type(0)) {
      return Type(1.0) / val;
    }
    return std::numeric_limits<Type>::max() + Type(1);
  }
};

template <>
struct UnaryFunc<ElwiseOpType::kReciprocal, half, half> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE half operator()(const half val) const {
    return half(1.0) / val;
  }
};

template <>
struct UnaryFunc<ElwiseOpType::kReciprocal, float, float> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE float operator()(const float val) const {
    return static_cast<float>(1.0) / val;
  }
};

template <>
struct UnaryFunc<ElwiseOpType::kReciprocal, double, double> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE double operator()(const double val) const {
    return static_cast<double>(1.0) / val;
  }
};

template <>
struct UnaryFunc<ElwiseOpType::kReciprocal, Complex<float>, Complex<float>> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Complex<float> operator()(const Complex<float> val) const {
    return Complex<float>(1.0) / val;
  }
};

template <>
struct UnaryFunc<ElwiseOpType::kReciprocal, Complex<double>, Complex<double>> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Complex<double> operator()(const Complex<double> val) const {
    return Complex<double>(1.0) / val;
  }
};
REGISTER_UNARY_OP_CUDA_FUNC_BOOL_TYPE(ElwiseOpType::kReciprocal);
REGISTER_UNARY_OP_CUDA_FUNC_INT_TYPE(ElwiseOpType::kReciprocal);
REGISTER_UNARY_OP_CUDA_FUNC_UINT_TYPE(ElwiseOpType::kReciprocal);
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kReciprocal);
REGISTER_UNARY_OP_CUDA_FUNC_COMPLEX_TYPE(ElwiseOpType::kReciprocal);

template <typename TypeIn, typename TypeOut>
struct UnaryFunc<ElwiseOpType::kReciprocal, TypeIn, TypeOut> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE TypeOut operator()(const TypeIn val) const {
    return TypeOut(1.0) / val;
  }
};
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, int64_t, float>(const size_t num,
    const int64_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, int, float>(const size_t num,
    const int *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, int16_t, float>(const size_t num,
    const int16_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, int8_t, float>(const size_t num,
    const int8_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, uint8_t, float>(const size_t num,
    const uint8_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, uint16_t, float>(const size_t num,
    const uint16_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, uint32_t, float>(const size_t num,
    const uint32_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, uint64_t, float>(const size_t num,
    const uint64_t *inp, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kReciprocal, bool, float>(const size_t num,
    const bool *inp, float *out, hipStream_t hip_stream);

template <typename Type>
struct UnaryFunc<ElwiseOpType::kExpm1, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return expm1(val); }
};
template <typename Type>
struct UnaryFunc<ElwiseOpType::kExpm1, Complex<Type>, Complex<Type>> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Complex<Type> operator()(const Complex<Type> val) const { return exp(val) - Complex<Type>(1.0); }
};
template <>
struct UnaryFunc<ElwiseOpType::kExpm1, float, float> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE float operator()(const float val) const { return expm1f(val); }
};
template <>
struct UnaryFunc<ElwiseOpType::kExpm1, half, half> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE half operator()(const half val) const { return __float2half(expm1f(__half2float(val))); }
};
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kExpm1);
REGISTER_UNARY_OP_CUDA_FUNC_COMPLEX_TYPE(ElwiseOpType::kExpm1);
template <typename Type>
struct UnaryFunc<ElwiseOpType::kOnesLike, Type, Type> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE Type operator()(const Type val) const { return Type(1.0); }
};
REGISTER_UNARY_OP_CUDA_FUNC_BOOL_TYPE(ElwiseOpType::kOnesLike);
REGISTER_UNARY_OP_CUDA_FUNC_INT_TYPE(ElwiseOpType::kOnesLike);
REGISTER_UNARY_OP_CUDA_FUNC_UINT_TYPE(ElwiseOpType::kOnesLike);
REGISTER_UNARY_OP_CUDA_FUNC_FLOAT_TYPE(ElwiseOpType::kOnesLike);
REGISTER_UNARY_OP_CUDA_FUNC_COMPLEX_TYPE(ElwiseOpType::kOnesLike);
template <typename Type>
struct UnaryFunc<ElwiseOpType::kLogicalNot, Type, bool> {
  DEVICE_HOST UnaryFunc() {}
  DEVICE bool operator()(const Type val) const { return !val; }
};
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, bool, bool>(const size_t num,
  const bool *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, int64_t, bool>(const size_t num,
  const int64_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, int, bool>(const size_t num,
  const int *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, int16_t, bool>(const size_t num,
  const int16_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, int8_t, bool>(const size_t num,
  const int8_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, uint8_t, bool>(const size_t num,
  const uint8_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, uint16_t, bool>(const size_t num,
  const uint16_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, uint32_t, bool>(const size_t num,
  const uint32_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, uint64_t, bool>(const size_t num,
  const uint64_t *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, half, bool>(const size_t num,
  const half *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, float, bool>(const size_t num,
  const float *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, double, bool>(const size_t num,
  const double *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, Complex<float>, bool>(
  const size_t num, const Complex<float> *inp, bool *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t UnaryOpsCudaFunc<ElwiseOpType::kLogicalNot, Complex<double>, bool>(
  const size_t num, const Complex<double> *inp, bool *out, hipStream_t hip_stream);

template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<ElwiseOpType::kZeta, In0_t, In1_t, Out_t> {
  DEVICE_HOST BinaryFunc() {}
  DEVICE Out_t operator()(const In0_t val0, const In1_t val1) const {
    return Eigen::internal::scalar_zeta_op<Out_t>()(val0, val1);
  }
};
template CUDA_LIB_EXPORT hipError_t BinaryOpsCudaFunc<ElwiseOpType::kZeta, float, float, float>(
  const size_t num, const float *in0, const float *in1, float *out, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BinaryOpsCudaFunc<ElwiseOpType::kZeta, double, double, double>(
  const size_t num, const double *in0, const double *in1, double *out, hipStream_t hip_stream);

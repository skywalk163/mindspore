#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/split_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "include/hip/hip_fp16.h"

template <typename T>
using Complex = mindspore::utils::Complex<T>;

template <typename T>
__global__ void Split(const size_t size, const int axis_step, const int all_size_before_axis, const int all_size_axis,
                      const T *input, T **outputs) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    int num = pos % all_size_before_axis / all_size_axis;
    int block = num / axis_step;
    int block_pos =
      pos / all_size_before_axis * axis_step * all_size_axis + num % axis_step * all_size_axis + pos % all_size_axis;
    outputs[block][block_pos] = input[pos];
  }
  return;
}

template <typename T>
hipError_t SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis, const int all_size_axis,
                        const T *input, T **outputs, hipStream_t hip_stream) {
  Split<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, axis_step, all_size_before_axis, all_size_axis, input,
                                                           outputs);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t SplitKernel<half>(const size_t size, const int axis_step,
                                                       const int all_size_before_axis, const int all_size_axis,
                                                       const half *input, half **outputs, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<float>(const size_t size, const int axis_step,
                                                        const int all_size_before_axis, const int all_size_axis,
                                                        const float *input, float **outputs, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<double>(const size_t size, const int axis_step,
                                                         const int all_size_before_axis, const int all_size_axis,
                                                         const double *input, double **outputs,
                                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<int8_t>(const size_t size, const int axis_step,
                                                         const int all_size_before_axis, const int all_size_axis,
                                                         const int8_t *input, int8_t **outputs,
                                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<int16_t>(const size_t size, const int axis_step,
                                                          const int all_size_before_axis, const int all_size_axis,
                                                          const int16_t *input, int16_t **outputs,
                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<int32_t>(const size_t size, const int axis_step,
                                                          const int all_size_before_axis, const int all_size_axis,
                                                          const int32_t *input, int32_t **outputs,
                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<int64_t>(const size_t size, const int axis_step,
                                                          const int all_size_before_axis, const int all_size_axis,
                                                          const int64_t *input, int64_t **outputs,
                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<uint8_t>(const size_t size, const int axis_step,
                                                          const int all_size_before_axis, const int all_size_axis,
                                                          const uint8_t *input, uint8_t **outputs,
                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<uint16_t>(const size_t size, const int axis_step,
                                                           const int all_size_before_axis, const int all_size_axis,
                                                           const uint16_t *input, uint16_t **outputs,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<uint32_t>(const size_t size, const int axis_step,
                                                           const int all_size_before_axis, const int all_size_axis,
                                                           const uint32_t *input, uint32_t **outputs,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<uint64_t>(const size_t size, const int axis_step,
                                                           const int all_size_before_axis, const int all_size_axis,
                                                           const uint64_t *input, uint64_t **outputs,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<Complex<float>>(const size_t size, const int axis_step,
                                                                 const int all_size_before_axis,
                                                                 const int all_size_axis, const Complex<float> *input,
                                                                 Complex<float> **outputs, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<Complex<double>>(const size_t size, const int axis_step,
                                                                  const int all_size_before_axis,
                                                                  const int all_size_axis, const Complex<double> *input,
                                                                  Complex<double> **outputs, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SplitKernel<bool>(const size_t size, const int axis_step,
                                                       const int all_size_before_axis, const int all_size_axis,
                                                       const bool *input, bool **outputs, hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_ops_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_common.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_pub_impl.cuh"

template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<BinaryOpType::kDiv, In0_t, In1_t, Out_t> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __forceinline__ Out_t operator()(In0_t val0, In1_t val1) const { return val0 / val1; }
};
REGISTER_BINARY_OP_CUDA_FUNC_COMPLEX_TYPE(BinaryOpType::kDiv);
REGISTER_BINARY_OP_CUDA_FUNC_FLOAT_TYPE(BinaryOpType::kDiv);
REGISTER_BINARY_OP_CUDA_FUNC_INT_TYPE(BinaryOpType::kDiv);

template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<BinaryOpType::kRealDiv, In0_t, In1_t, Out_t> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ Out_t operator()(const In0_t &lhs, const In1_t &rhs) const { return (lhs / rhs); }
};
REGISTER_BINARY_OP_CUDA_FUNC_COMPLEX_TYPE(BinaryOpType::kRealDiv);
REGISTER_BINARY_OP_CUDA_FUNC_FLOAT_TYPE(BinaryOpType::kRealDiv);
REGISTER_BINARY_OP_CUDA_FUNC_INT_TYPE(BinaryOpType::kRealDiv);

// DivNoNan check if rhs is less than epsilon
template <typename T>
struct BinaryFunc<BinaryOpType::kDivNoNan, T, T, T, typename std::is_floating_point<T>::type> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  // default T is float
  __device__ __host__ __forceinline__ T operator()(const T &lhs, const T &rhs) const {
    return rhs < Eps<T>() && rhs > -Eps<T>() ? 0.0 : (lhs / rhs);
  }
};
template <typename T>
struct BinaryFunc<BinaryOpType::kDivNoNan, T, T, T, typename std::is_integral<T>::type> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ int operator()(const T &lhs, const T &rhs) const {
    return rhs == 0 ? 0 : (lhs / rhs);
  }
};
template <>
struct BinaryFunc<BinaryOpType::kDivNoNan, half, half, half> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ half operator()(const half &lhs, const half &rhs) const {
    if (__half2float(rhs) < (0.00001) && __half2float(rhs) > -0.00001) {
      return static_cast<half>(0.0);
    }
    return __float2half_rn(__half2float(lhs) / __half2float(rhs));
  }
};
template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<BinaryOpType::kDivNoNan, In0_t, In1_t, Complex<Out_t>> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ Complex<Out_t> operator()(const In0_t &lhs, const In1_t &rhs) const {
    Complex<Out_t> complex_rhs(rhs);
    if ((complex_rhs.real() < Eps<float>() && complex_rhs.real() > -Eps<float>()) ||
        (complex_rhs.imag() < Eps<float>() && complex_rhs.imag() > -Eps<float>())) {
      Complex<Out_t> res(0.0, 0.0);
      return res;
    }
    return lhs / rhs;
  }
};
REGISTER_BINARY_OP_CUDA_FUNC_COMPLEX_TYPE(BinaryOpType::kDivNoNan);
REGISTER_BINARY_OP_CUDA_FUNC_FLOAT_TYPE(BinaryOpType::kDivNoNan);
REGISTER_BINARY_OP_CUDA_FUNC_INT_TYPE(BinaryOpType::kDivNoNan);

#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITH WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/broadcast_to_impl.cuh"
#include <math.h>
#include <vector>
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"

// copy
template <typename T>
__global__ void BroadcastToCpyCuda(size_t dim_size, size_t output_num, UnaryBroadcastStrideInfo strides, T *input,
                                   T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    int64_t cur_out_idx = 0;
    size_t cur_pos = pos;
    size_t inp_pos = 0;
    for (int idx = 0; idx < dim_size; ++idx) {
      cur_out_idx = cur_pos / strides.output_stride[idx];
      inp_pos += cur_out_idx * strides.input_stride[idx];
      cur_pos -= cur_out_idx * strides.output_stride[idx];
    }
    output[pos] = input[inp_pos];
  }
}

UnaryBroadcastStrideInfo UnaryBroadcastCalStride(const size_t dim_size, const std::vector<int64_t> &inp_shape,
                                                 const std::vector<int64_t> &out_shape) {
  UnaryBroadcastStrideInfo strides;
  strides.input_stride[dim_size - 1] = 1;
  strides.output_stride[dim_size - 1] = 1;
  for (int64_t idx = dim_size - 2; idx >= 0; --idx) {
    strides.output_stride[idx] = out_shape[idx + 1] * strides.output_stride[idx + 1];
    strides.input_stride[idx] = inp_shape[idx + 1] * strides.input_stride[idx + 1];
  }
  for (size_t idx = 0; idx < dim_size; ++idx) {
    strides.input_stride[idx] = (inp_shape[idx] == 1) ? 0 : strides.input_stride[idx];
  }
  return strides;
}

BinaryBroadcastStrideInfo BinaryBroadcastCalStride(const size_t dim_size, const std::vector<int64_t> &in0_shape,
                                                   const std::vector<int64_t> &in1_shape,
                                                   const std::vector<int64_t> &out_shape) {
  BinaryBroadcastStrideInfo strides;
  strides.in0_stride[dim_size - 1] = 1;
  strides.in1_stride[dim_size - 1] = 1;
  strides.out_stride[dim_size - 1] = 1;
  for (int64_t idx = dim_size - 2; idx >= 0; --idx) {
    strides.out_stride[idx] = out_shape[idx + 1] * strides.out_stride[idx + 1];
    strides.in0_stride[idx] = in0_shape[idx + 1] * strides.in0_stride[idx + 1];
    strides.in1_stride[idx] = in1_shape[idx + 1] * strides.in1_stride[idx + 1];
  }
  for (size_t idx = 0; idx < dim_size; ++idx) {
    strides.in0_stride[idx] = (in0_shape[idx] == 1) ? 0 : strides.in0_stride[idx];
    strides.in1_stride[idx] = (in1_shape[idx] == 1) ? 0 : strides.in1_stride[idx];
  }
  return strides;
}

TrinaryBroadcastStrideInfo TrinaryBroadcastCalStride(const size_t dim_size, const std::vector<int64_t> &in0_shape,
                                                     const std::vector<int64_t> &in1_shape,
                                                     const std::vector<int64_t> &in2_shape,
                                                     const std::vector<int64_t> &out_shape) {
  TrinaryBroadcastStrideInfo strides;
  strides.in0_stride[dim_size - 1] = 1;
  strides.in1_stride[dim_size - 1] = 1;
  strides.in2_stride[dim_size - 1] = 1;
  strides.out_stride[dim_size - 1] = 1;
  for (int64_t idx = dim_size - 2; idx >= 0; --idx) {
    strides.out_stride[idx] = out_shape[idx + 1] * strides.out_stride[idx + 1];
    strides.in0_stride[idx] = in0_shape[idx + 1] * strides.in0_stride[idx + 1];
    strides.in1_stride[idx] = in1_shape[idx + 1] * strides.in1_stride[idx + 1];
    strides.in2_stride[idx] = in2_shape[idx + 1] * strides.in2_stride[idx + 1];
  }
  for (size_t idx = 0; idx < dim_size; ++idx) {
    strides.in0_stride[idx] = (in0_shape[idx] == 1) ? 0 : strides.in0_stride[idx];
    strides.in1_stride[idx] = (in1_shape[idx] == 1) ? 0 : strides.in1_stride[idx];
    strides.in2_stride[idx] = (in2_shape[idx] == 1) ? 0 : strides.in2_stride[idx];
  }
  return strides;
}

template <typename T>
hipError_t BroadcastTo(const std::vector<int64_t> &inp_shape, const std::vector<int64_t> &out_shape, T *input,
                        T *output, size_t device_id, hipStream_t hip_stream) {
  const size_t dim_size = out_shape.size();
  size_t output_num = 1;
  for (auto val : out_shape) {
    output_num *= val;
  }
  UnaryBroadcastStrideInfo strides = UnaryBroadcastCalStride(dim_size, inp_shape, out_shape);
  size_t thread_num = output_num > 1024 ? 1024 : output_num;
  BroadcastToCpyCuda<T><<<CUDA_BLOCKS_CAL(device_id, output_num, thread_num), thread_num, 0, hip_stream>>>(
    dim_size, output_num, strides, input, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t BroadcastTo<bool>(const std::vector<int64_t> &inp_shape,
                                                       const std::vector<int64_t> &out_shape, bool *input, bool *output,
                                                       size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<int8_t>(const std::vector<int64_t> &inp_shape,
                                                         const std::vector<int64_t> &out_shape, int8_t *input,
                                                         int8_t *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<int16_t>(const std::vector<int64_t> &inp_shape,
                                                          const std::vector<int64_t> &out_shape, int16_t *input,
                                                          int16_t *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<int32_t>(const std::vector<int64_t> &inp_shape,
                                                          const std::vector<int64_t> &out_shape, int32_t *input,
                                                          int32_t *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<int64_t>(const std::vector<int64_t> &inp_shape,
                                                          const std::vector<int64_t> &out_shape, int64_t *input,
                                                          int64_t *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<uint8_t>(const std::vector<int64_t> &inp_shape,
                                                          const std::vector<int64_t> &out_shape, uint8_t *input,
                                                          uint8_t *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<uint16_t>(const std::vector<int64_t> &inp_shape,
                                                           const std::vector<int64_t> &out_shape, uint16_t *input,
                                                           uint16_t *output, size_t device_id,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<uint32_t>(const std::vector<int64_t> &inp_shape,
                                                           const std::vector<int64_t> &out_shape, uint32_t *input,
                                                           uint32_t *output, size_t device_id,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<uint64_t>(const std::vector<int64_t> &inp_shape,
                                                           const std::vector<int64_t> &out_shape, uint64_t *input,
                                                           uint64_t *output, size_t device_id,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<half>(const std::vector<int64_t> &inp_shape,
                                                       const std::vector<int64_t> &out_shape, half *input, half *output,
                                                       size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<float>(const std::vector<int64_t> &inp_shape,
                                                        const std::vector<int64_t> &out_shape, float *input,
                                                        float *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<double>(const std::vector<int64_t> &inp_shape,
                                                         const std::vector<int64_t> &out_shape, double *input,
                                                         double *output, size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<Complex<float>>(const std::vector<int64_t> &inp_shape,
                                                                 const std::vector<int64_t> &out_shape,
                                                                 Complex<float> *input, Complex<float> *output,
                                                                 size_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastTo<Complex<double>>(const std::vector<int64_t> &inp_shape,
                                                                  const std::vector<int64_t> &out_shape,
                                                                  Complex<double> *input, Complex<double> *output,
                                                                  size_t device_id, hipStream_t hip_stream);

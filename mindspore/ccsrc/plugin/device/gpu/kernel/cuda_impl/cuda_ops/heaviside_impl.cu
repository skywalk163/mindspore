#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/heaviside_impl.cuh"

__constant__ size_t start_cal[5];
__constant__ size_t end_cal[5];
__constant__ size_t output_cal[5];

template <typename T>
struct HeavisideFunc {
  __device__ __host__ __forceinline__ T operator()(const T &x1, const T &x2) {
    if (x1 > T(0)) {
      return T(1);
    } else if (x1 == T(0)) {
      return x2;
    } else {
      return T(0);
    }
  }
};

template <typename T, typename Func>
__global__ void CalHeavisideKernel(size_t size, const T *x1, const T *x2, T *y) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    y[pos] = Func()(x1[pos], x2[pos]);
  }
}

__device__ __forceinline__ size_t Index(const size_t &index, const size_t &dim) { return dim == 1 ? 0 : index; }

template <typename T, typename Func>
__global__ void BroadcastHeavisideKernel(const size_t l0, const size_t l1, const size_t l2, const size_t l3,
                                         const size_t l4, const size_t l5, const size_t l6, const size_t r0,
                                         const size_t r1, const size_t r2, const size_t r3, const size_t r4,
                                         const size_t r5, const size_t r6, const size_t d0, const size_t d1,
                                         const size_t d2, const size_t d3, const size_t d4, const size_t d5,
                                         const size_t d6, const T *x1, const T *x2, T *y) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < d0 * d1 * d2 * d3 * d4 * d5 * d6;
       pos += blockDim.x * gridDim.x) {
    size_t i = pos / output_cal[0] % d0;
    size_t j = pos / output_cal[1] % d1;
    size_t k = pos / output_cal[2] % d2;
    size_t l = pos / output_cal[3] % d3;
    size_t m = pos / output_cal[4] % d4;
    size_t n = pos / d6 % d5;
    size_t o = pos % d6;

    size_t l_index = Index(i, l0) * start_cal[0];
    l_index += Index(j, l1) * start_cal[1];
    l_index += Index(k, l2) * start_cal[2];
    l_index += Index(l, l3) * start_cal[3];
    l_index += Index(m, l4) * start_cal[4];
    l_index += Index(n, l5) * l6;
    l_index += Index(o, l6);
    size_t r_index = Index(i, r0) * end_cal[0];
    r_index += Index(j, r1) * end_cal[1];
    r_index += Index(k, r2) * end_cal[2];
    r_index += Index(l, r3) * end_cal[3];
    r_index += Index(m, r4) * end_cal[4];
    r_index += Index(n, r5) * r6;
    r_index += Index(o, r6);
    y[pos] = Func()(x1[l_index], x2[r_index]);
  }
}

template <typename T>
hipError_t CalHeaviside(size_t size, const T *x1, const T *x2, T *y, const uint32_t &device_id,
                         hipStream_t hip_stream) {
  CalHeavisideKernel<T, HeavisideFunc<T>>
    <<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, x1, x2, y);
  return GetCudaStatus();
}

hipError_t CalData(const std::vector<size_t> &start_shape, size_t *output) {
  output[4] = start_shape[5] * start_shape[6];
  output[3] = output[4] * start_shape[4];
  output[2] = output[3] * start_shape[3];
  output[1] = output[2] * start_shape[2];
  output[0] = output[1] * start_shape[1];
  return GetCudaStatus();
}

template <typename T>
hipError_t BroadcastHeaviside(const std::vector<size_t> &x1_shape, const std::vector<size_t> &x2_shape,
                               const std::vector<size_t> &y_shape, const T *x1, const T *x2, T *y,
                               const uint32_t &device_id, hipStream_t hip_stream) {
  size_t size = 1;
  for (auto d : y_shape) {
    size *= d;
  }
  size_t start_dim[5];
  size_t end_dim[5];
  size_t output_dim[5];
  CalData(x1_shape, start_dim);
  CalData(x2_shape, end_dim);
  CalData(y_shape, output_dim);
  hipMemcpyToSymbol(HIP_SYMBOL(start_cal), start_dim, sizeof(size_t) * 5);
  hipMemcpyToSymbol(HIP_SYMBOL(end_cal), end_dim, sizeof(size_t) * 5);
  hipMemcpyToSymbol(HIP_SYMBOL(output_cal), output_dim, sizeof(size_t) * 5);
  BroadcastHeavisideKernel<T, HeavisideFunc<T>>
    <<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(
      x1_shape[0], x1_shape[1], x1_shape[2], x1_shape[3], x1_shape[4], x1_shape[5], x1_shape[6], x2_shape[0],
      x2_shape[1], x2_shape[2], x2_shape[3], x2_shape[4], x2_shape[5], x2_shape[6], y_shape[0], y_shape[1], y_shape[2],
      y_shape[3], y_shape[4], y_shape[5], y_shape[6], x1, x2, y);
  return GetCudaStatus();
}
template CUDA_LIB_EXPORT hipError_t CalHeaviside<uint8_t>(size_t, const uint8_t *, const uint8_t *, uint8_t *,
                                                           const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<uint16_t>(size_t, const uint16_t *, const uint16_t *, uint16_t *,
                                                            const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<uint32_t>(size_t, const uint32_t *, const uint32_t *, uint32_t *,
                                                            const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<uint64_t>(size_t, const uint64_t *, const uint64_t *, uint64_t *,
                                                            const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<int8_t>(size_t, const int8_t *, const int8_t *, int8_t *,
                                                          const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<int16_t>(size_t, const int16_t *, const int16_t *, int16_t *,
                                                           const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<int32_t>(size_t, const int32_t *, const int32_t *, int32_t *,
                                                           const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<int64_t>(size_t, const int64_t *, const int64_t *, int64_t *,
                                                           const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<half>(size_t, const half *, const half *, half *, const uint32_t &,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<float>(size_t, const float *, const float *, float *,
                                                         const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalHeaviside<double>(size_t, const double *, const double *, double *,
                                                          const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<uint8_t>(const std::vector<size_t> &,
                                                                 const std::vector<size_t> &,
                                                                 const std::vector<size_t> &, const uint8_t *,
                                                                 const uint8_t *, uint8_t *, const uint32_t &,
                                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<uint16_t>(const std::vector<size_t> &,
                                                                  const std::vector<size_t> &,
                                                                  const std::vector<size_t> &, const uint16_t *,
                                                                  const uint16_t *, uint16_t *, const uint32_t &,
                                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<uint32_t>(const std::vector<size_t> &,
                                                                  const std::vector<size_t> &,
                                                                  const std::vector<size_t> &, const uint32_t *,
                                                                  const uint32_t *, uint32_t *, const uint32_t &,
                                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<uint64_t>(const std::vector<size_t> &,
                                                                  const std::vector<size_t> &,
                                                                  const std::vector<size_t> &, const uint64_t *,
                                                                  const uint64_t *, uint64_t *, const uint32_t &,
                                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<int8_t>(const std::vector<size_t> &,
                                                                const std::vector<size_t> &,
                                                                const std::vector<size_t> &, const int8_t *,
                                                                const int8_t *, int8_t *, const uint32_t &,
                                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<int16_t>(const std::vector<size_t> &,
                                                                 const std::vector<size_t> &,
                                                                 const std::vector<size_t> &, const int16_t *,
                                                                 const int16_t *, int16_t *, const uint32_t &,
                                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<int32_t>(const std::vector<size_t> &,
                                                                 const std::vector<size_t> &,
                                                                 const std::vector<size_t> &, const int32_t *,
                                                                 const int32_t *, int32_t *, const uint32_t &,
                                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<int64_t>(const std::vector<size_t> &,
                                                                 const std::vector<size_t> &,
                                                                 const std::vector<size_t> &, const int64_t *,
                                                                 const int64_t *, int64_t *, const uint32_t &,
                                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<half>(const std::vector<size_t> &, const std::vector<size_t> &,
                                                              const std::vector<size_t> &, const half *, const half *,
                                                              half *, const uint32_t &, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<float>(const std::vector<size_t> &, const std::vector<size_t> &,
                                                               const std::vector<size_t> &, const float *,
                                                               const float *, float *, const uint32_t &,
                                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t BroadcastHeaviside<double>(const std::vector<size_t> &,
                                                                const std::vector<size_t> &,
                                                                const std::vector<size_t> &, const double *,
                                                                const double *, double *, const uint32_t &,
                                                                hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/fft_with_size_impl.cuh"
#include "include/hip/hip_runtime.h"

// cublas exec scale
#ifndef CUBLAS_EXEC_SCALE
#define CUBLAS_EXEC_SCALE(cublas_exec, real, cu_complex)                \
  do {                                                                  \
    if (scale_factor != 1.0) {                                          \
      auto alpha = static_cast<real>(scale_factor);                     \
      auto out = static_cast<cu_complex *>(y_ptr);                      \
      CUBLAS_CALL(cublas_exec(scale_plan, y_elements, &alpha, out, 1)); \
    }                                                                   \
  } while (0)
#endif  // CUBLAS_EXEC_SCALE

hipError_t CalculateFFT(hipfftComplex *x_ptr, hipfftComplex *y_ptr, const double &scale_factor, const int &y_elements,
                         hipfftHandle cufft_plan, hipblasHandle_t scale_plan, const uint32_t &device_id,
                         hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  CUFFT_CALL(hipfftExecC2C(cufft_plan, x_ptr, y_ptr, HIPFFT_FORWARD));
  CUBLAS_EXEC_SCALE(hipblasCsscal, float, hipComplex);
  return GetCudaStatus();
}

hipError_t CalculateFFT(hipfftDoubleComplex *x_ptr, hipfftDoubleComplex *y_ptr, const double &scale_factor,
                         const int &y_elements, hipfftHandle cufft_plan, hipblasHandle_t scale_plan,
                         const uint32_t &device_id, hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  CUFFT_CALL(hipfftExecZ2Z(cufft_plan, x_ptr, y_ptr, HIPFFT_FORWARD));
  CUBLAS_EXEC_SCALE(hipblasZdscal, double, hipDoubleComplex);
  return GetCudaStatus();
}

hipError_t CalculateIFFT(hipfftComplex *x_ptr, hipfftComplex *y_ptr, const double &scale_factor, const int &y_elements,
                          hipfftHandle cufft_plan, hipblasHandle_t scale_plan, const uint32_t &device_id,
                          hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  CUFFT_CALL(hipfftExecC2C(cufft_plan, x_ptr, y_ptr, HIPFFT_BACKWARD));
  CUBLAS_EXEC_SCALE(hipblasCsscal, float, hipComplex);
  return GetCudaStatus();
}

hipError_t CalculateIFFT(hipfftDoubleComplex *x_ptr, hipfftDoubleComplex *y_ptr, const double &scale_factor,
                          const int &y_elements, hipfftHandle cufft_plan, hipblasHandle_t scale_plan,
                          const uint32_t &device_id, hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  CUFFT_CALL(hipfftExecZ2Z(cufft_plan, x_ptr, y_ptr, HIPFFT_BACKWARD));
  CUBLAS_EXEC_SCALE(hipblasZdscal, double, hipDoubleComplex);
  return GetCudaStatus();
}

__global__ void Float2FloatComplex(const float *input_addr, hipfftComplex *output_addr, const int len) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = make_hipFloatComplex(input_addr[pos], 0.);
  }
}

__global__ void Double2DoubleComplex(const double *input_addr, hipfftDoubleComplex *output_addr, const int len) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = make_hipDoubleComplex(input_addr[pos], 0.);
  }
}

__global__ void FloatComplex2Float(const hipfftComplex *input_addr, float *output_addr, const int len) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = hipCrealf(input_addr[pos]);
  }
}

__global__ void DoubleComplex2Double(const hipfftDoubleComplex *input_addr, double *output_addr, const int len) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = hipCreal(input_addr[pos]);
  }
}

hipError_t CalculateRFFT(float *x_ptr, hipfftComplex *w_ptr, hipfftComplex *y_ptr, const bool &is_onesided,
                          const double &scale_factor, const int &x_elements, const int &y_elements,
                          hipfftHandle cufft_plan, hipblasHandle_t scale_plan, const uint32_t &device_id,
                          hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  if (is_onesided) {  // onesided use native cufft r2c
    CUFFT_CALL(hipfftExecR2C(cufft_plan, x_ptr, y_ptr));
  } else {  // full freq use [casting + c2c], cast real input buffer to complex workspace buffer
    Float2FloatComplex<<<CUDA_BLOCKS(device_id, x_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(x_ptr, w_ptr,
                                                                                                        x_elements);
    CUFFT_CALL(hipfftExecC2C(cufft_plan, w_ptr, y_ptr, HIPFFT_FORWARD));
  }
  CUBLAS_EXEC_SCALE(hipblasCsscal, float, hipComplex);
  return GetCudaStatus();
}

hipError_t CalculateRFFT(double *x_ptr, hipfftDoubleComplex *w_ptr, hipfftDoubleComplex *y_ptr, const bool &is_onesided,
                          const double &scale_factor, const int &x_elements, const int &y_elements,
                          hipfftHandle cufft_plan, hipblasHandle_t scale_plan, const uint32_t &device_id,
                          hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  if (is_onesided) {  // onesided use native cufft r2c
    CUFFT_CALL(hipfftExecD2Z(cufft_plan, x_ptr, y_ptr));
  } else {  // full freq use [casting + c2c], cast real input buffer to complex workspace buffer
    Double2DoubleComplex<<<CUDA_BLOCKS(device_id, x_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(x_ptr, w_ptr,
                                                                                                          x_elements);
    CUFFT_CALL(hipfftExecZ2Z(cufft_plan, w_ptr, y_ptr, HIPFFT_FORWARD));
  }
  CUBLAS_EXEC_SCALE(hipblasZdscal, double, hipDoubleComplex);
  return GetCudaStatus();
}

hipError_t CalculateIRFFT(hipfftComplex *x_ptr, hipfftComplex *w_ptr, float *y_ptr, const bool &is_onesided,
                    const double &scale_factor, const int &x_elements, const int &y_elements, hipfftHandle cufft_plan,
                    hipblasHandle_t scale_plan, const uint32_t &device_id, hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  if (is_onesided) {  // onesided use native cufft c2r
    // complex-to-real need to copy input buffer to tmp buffer to avoid cufft overwriting.
    CUDA_RT_CALL(hipMemcpyAsync(w_ptr, x_ptr, x_elements * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
    CUFFT_CALL(hipfftExecC2R(cufft_plan, w_ptr, y_ptr));
  } else {  // full freq use [c2c + casting]
    // dump c2c result to workspace buffer, then cast complex workspace buffer to real output buffer.
    CUFFT_CALL(hipfftExecC2C(cufft_plan, x_ptr, w_ptr, HIPFFT_BACKWARD));
    FloatComplex2Float<<<CUDA_BLOCKS(device_id, y_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(w_ptr, y_ptr,
                                                                                                        y_elements);
  }
  CUBLAS_EXEC_SCALE(hipblasSscal, float, float);
  return GetCudaStatus();
}

hipError_t CalculateIRFFT(hipfftDoubleComplex *x_ptr, hipfftDoubleComplex *w_ptr, double *y_ptr, const bool &is_onesided,
                           const double &scale_factor, const int &x_elements, const int &y_elements,
                           hipfftHandle cufft_plan, hipblasHandle_t scale_plan, const uint32_t &device_id,
                           hipStream_t hip_stream) {
  CUFFT_CALL(hipfftSetStream(cufft_plan, hip_stream));
  CUBLAS_CALL(hipblasSetStream(scale_plan, hip_stream));
  if (is_onesided) {  // onesided use native cufft r2c
    // complex-to-real need to copy input buffer to tmp buffer to avoid cufft overwriting.
    CUDA_RT_CALL(hipMemcpyAsync(w_ptr, x_ptr, x_elements * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToDevice));
    CUFFT_CALL(hipfftExecZ2D(cufft_plan, w_ptr, y_ptr));
  } else {  // full freq use [c2c + casting]
    // dump c2c result to workspace buffer, then cast complex workspace buffer to real output buffer.
    CUFFT_CALL(hipfftExecZ2Z(cufft_plan, x_ptr, w_ptr, HIPFFT_BACKWARD));
    DoubleComplex2Double<<<CUDA_BLOCKS(device_id, y_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(w_ptr, y_ptr,
                                                                                                          y_elements);
  }
  CUBLAS_EXEC_SCALE(hipblasDscal, double, double);
  return GetCudaStatus();
}

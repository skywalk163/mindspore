#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/index_fill_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"

template <typename DataType, typename Int>
__global__ void IndexFillKernel(const int *__restrict__ index_ptr, const DataType *__restrict__ value_ptr,
                                bool *__restrict__ out_bound_ptr, DataType *__restrict__ out_ptr, Int dim_size,
                                Int inner_size, Int outer_inner_size, Int index_num) {
  DataType fill_value = *value_ptr;
  Int start_idx = static_cast<Int>(blockIdx.x * blockDim.x + threadIdx.x);
  Int step = static_cast<Int>(blockDim.x * gridDim.x);
  for (Int tid = start_idx; tid < index_num; tid += step) {
    Int index_idx = tid / outer_inner_size;
    Int outer_inner_idx = tid % outer_inner_size;
    // Each index must be [-dim_size, dim_size)
    Int dim_idx = static_cast<Int>(index_ptr[index_idx]);
    if (dim_idx < -dim_size || dim_idx >= dim_size) {
      *out_bound_ptr = true;
      break;
    } else if (dim_idx < 0) {
      dim_idx += dim_size;
    }
    Int inner_idx = outer_inner_idx % inner_size;
    Int outer_idx = (outer_inner_idx - inner_idx) * dim_size;
    Int out_idx = outer_idx + dim_idx * inner_size + inner_idx;
    out_ptr[out_idx] = fill_value;
  }
}

template <typename DataType>
hipError_t IndexFill(DataType *out_ptr, const int *index_ptr, int64_t index_size, int64_t outer_size, int64_t dim_size,
                      int64_t inner_size, const DataType *value_ptr, bool *out_bound_ptr, const uint32_t &device_id,
                      hipStream_t hip_stream) {
  int64_t outer_inner_size = outer_size * inner_size;
  int64_t index_num = outer_inner_size * index_size;
  int64_t element_num = outer_inner_size * dim_size;
  int64_t max_int32_value = std::numeric_limits<int>::max();
  auto grids = CUDA_BLOCKS(device_id, index_num);
  auto blocks = CUDA_THREADS(device_id);
  if (index_num <= max_int32_value && element_num <= max_int32_value) {
    IndexFillKernel<DataType, int><<<grids, blocks, 0, hip_stream>>>(
      index_ptr, value_ptr, out_bound_ptr, out_ptr, static_cast<int>(dim_size), static_cast<int>(inner_size),
      static_cast<int>(outer_inner_size), static_cast<int>(index_num));
  } else {
    IndexFillKernel<DataType, int64_t><<<grids, blocks, 0, hip_stream>>>(
      index_ptr, value_ptr, out_bound_ptr, out_ptr, dim_size, inner_size, outer_inner_size, index_num);
  }
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t IndexFill<bool>(bool *out_ptr, const int *index_ptr, int64_t index_size,
                                                     int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                     const bool *value_ptr, bool *out_bound_ptr,
                                                     const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<uint8_t>(uint8_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                        int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                        const uint8_t *value_ptr, bool *out_bound_ptr,
                                                        const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<uint16_t>(uint16_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                         int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                         const uint16_t *value_ptr, bool *out_bound_ptr,
                                                         const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<uint32_t>(uint32_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                         int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                         const uint32_t *value_ptr, bool *out_bound_ptr,
                                                         const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<uint64_t>(uint64_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                         int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                         const uint64_t *value_ptr, bool *out_bound_ptr,
                                                         const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<int8_t>(int8_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                       int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                       const int8_t *value_ptr, bool *out_bound_ptr,
                                                       const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<int16_t>(int16_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                        int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                        const int16_t *value_ptr, bool *out_bound_ptr,
                                                        const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<int32_t>(int32_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                        int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                        const int32_t *value_ptr, bool *out_bound_ptr,
                                                        const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<int64_t>(int64_t *out_ptr, const int *index_ptr, int64_t index_size,
                                                        int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                        const int64_t *value_ptr, bool *out_bound_ptr,
                                                        const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<half>(half *out_ptr, const int *index_ptr, int64_t index_size,
                                                     int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                     const half *value_ptr, bool *out_bound_ptr,
                                                     const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<float>(float *out_ptr, const int *index_ptr, int64_t index_size,
                                                      int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                      const float *value_ptr, bool *out_bound_ptr,
                                                      const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<double>(double *out_ptr, const int *index_ptr, int64_t index_size,
                                                       int64_t outer_size, int64_t dim_size, int64_t inner_size,
                                                       const double *value_ptr, bool *out_bound_ptr,
                                                       const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<Complex<float>>(Complex<float> *out_ptr, const int *index_ptr,
                                                               int64_t index_size, int64_t outer_size, int64_t dim_size,
                                                               int64_t inner_size, const Complex<float> *value_ptr,
                                                               bool *out_bound_ptr, const uint32_t &device_id,
                                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t IndexFill<Complex<double>>(Complex<double> *out_ptr, const int *index_ptr,
                                                                int64_t index_size, int64_t outer_size,
                                                                int64_t dim_size, int64_t inner_size,
                                                                const Complex<double> *value_ptr, bool *out_bound_ptr,
                                                                const uint32_t &device_id, hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/embedding_lookup_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T, typename S>
__global__ void EmbeddingLookupKernel(T *input, S *indices, T *output, size_t output_dim0, size_t output_dim1,
                                      size_t output_dim2, size_t input_dim1, int64_t offset) {
  size_t size = output_dim0 * output_dim1 * output_dim2;
  size_t i, j;
  for (size_t write_idx = blockIdx.x * blockDim.x + threadIdx.x; write_idx < size;
       write_idx += blockDim.x * gridDim.x) {
    i = write_idx / output_dim2 % output_dim1;
    j = write_idx % output_dim2;

    S index_after_offset = indices[i] - static_cast<S>(offset);
    if ((index_after_offset >= 0) && (index_after_offset < input_dim1)) {
      size_t read_idx = index_after_offset * output_dim2 + j;
      output[write_idx] = input[read_idx];
    } else {
      output[write_idx] = 0;
    }
  }

  return;
}

template <typename T, typename S>
hipError_t CalEmbeddingLookup(T *input, S *indices, T *output, size_t output_dim0, size_t output_dim1,
                               size_t output_dim2, size_t input_dim1, int64_t offset, hipStream_t stream) {
  size_t size = output_dim0 * output_dim1 * output_dim2;
  EmbeddingLookupKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, indices, output, output_dim0, output_dim1,
                                                                      output_dim2, input_dim1, offset);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<float, int>(float *input, int *indices, float *output,
                                                                    size_t output_dim0, size_t output_dim1,
                                                                    size_t output_dim2, size_t input_dim1,
                                                                    int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<float, int64_t>(float *input, int64_t *indices, float *output,
                                                                        size_t output_dim0, size_t output_dim1,
                                                                        size_t output_dim2, size_t input_dim1,
                                                                        int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<half, int>(half *input, int *indices, half *output,
                                                                   size_t output_dim0, size_t output_dim1,
                                                                   size_t output_dim2, size_t input_dim1,
                                                                   int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<half, int64_t>(half *input, int64_t *indices, half *output,
                                                                       size_t output_dim0, size_t output_dim1,
                                                                       size_t output_dim2, size_t input_dim1,
                                                                       int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<double, int>(double *input, int *indices, double *output,
                                                                     size_t output_dim0, size_t output_dim1,
                                                                     size_t output_dim2, size_t input_dim1,
                                                                     int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<double, int64_t>(double *input, int64_t *indices,
                                                                         double *output, size_t output_dim0,
                                                                         size_t output_dim1, size_t output_dim2,
                                                                         size_t input_dim1, int64_t offset,
                                                                         hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int, int>(int *input, int *indices, int *output,
                                                                  size_t output_dim0, size_t output_dim1,
                                                                  size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int, int64_t>(int *input, int64_t *indices, int *output,
                                                                      size_t output_dim0, size_t output_dim1,
                                                                      size_t output_dim2, size_t input_dim1,
                                                                      int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int16_t, int>(int16_t *input, int *indices, int16_t *output,
                                                                      size_t output_dim0, size_t output_dim1,
                                                                      size_t output_dim2, size_t input_dim1,
                                                                      int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int16_t, int64_t>(int16_t *input, int64_t *indices,
                                                                          int16_t *output, size_t output_dim0,
                                                                          size_t output_dim1, size_t output_dim2,
                                                                          size_t input_dim1, int64_t offset,
                                                                          hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int8_t, int>(int8_t *input, int *indices, int8_t *output,
                                                                     size_t output_dim0, size_t output_dim1,
                                                                     size_t output_dim2, size_t input_dim1,
                                                                     int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<int8_t, int64_t>(int8_t *input, int64_t *indices,
                                                                         int8_t *output, size_t output_dim0,
                                                                         size_t output_dim1, size_t output_dim2,
                                                                         size_t input_dim1, int64_t offset,
                                                                         hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<uint8_t, int>(uint8_t *input, int *indices, uint8_t *output,
                                                                      size_t output_dim0, size_t output_dim1,
                                                                      size_t output_dim2, size_t input_dim1,
                                                                      int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<uint8_t, int64_t>(uint8_t *input, int64_t *indices,
                                                                          uint8_t *output, size_t output_dim0,
                                                                          size_t output_dim1, size_t output_dim2,
                                                                          size_t input_dim1, int64_t offset,
                                                                          hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<bool, int>(bool *input, int *indices, bool *output,
                                                                   size_t output_dim0, size_t output_dim1,
                                                                   size_t output_dim2, size_t input_dim1,
                                                                   int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t CalEmbeddingLookup<bool, int64_t>(bool *input, int64_t *indices, bool *output,
                                                                       size_t output_dim0, size_t output_dim1,
                                                                       size_t output_dim2, size_t input_dim1,
                                                                       int64_t offset, hipStream_t stream);

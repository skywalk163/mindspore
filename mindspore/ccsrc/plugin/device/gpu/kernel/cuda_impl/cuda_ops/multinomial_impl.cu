#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "multinomial_impl.cuh"
#include <algorithm>

template <typename T, typename S>
inline T Floor(const T &num, const S &unit) {
  return static_cast<T>(num / unit);
}

template <typename T, typename S>
inline T Ceil(const T &num, const S &unit) {
  return static_cast<T>((num + unit - 1) / unit);
}

__global__ void InitRandStateKernel(uint64_t seed, uint64_t seed_offset, int num, hiprandState *state) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, seed_offset, &state[i]);
  }
}

hipError_t InitRandState(uint64_t seed, uint64_t seed_offset, int num, hiprandState *state, hipStream_t stream) {
  InitRandStateKernel<<<(num + 127) / 128, 128, 0, stream>>>(seed, seed_offset, num, state);
  return GetCudaStatus();
}

template <typename T>
__global__ void CheckZeroKernel(const size_t distributions, const size_t categories, const T *input, T *out) {
  out[0] = 0;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (distributions); pos += blockDim.x * gridDim.x) {
    if (input[(1 + pos) * categories - 1] <= 0) {
      out[0] = 1;
    }
  }
  return;
}

template <typename T>
hipError_t CheckZero(const size_t distributions, const size_t categories, const T *input, T *output,
                      hipStream_t hip_stream) {
  CheckZeroKernel<<<GET_BLOCKS(distributions), GET_THREADS, 0, hip_stream>>>(distributions, categories, input, output);
  return GetCudaStatus();
}

template <typename T>
__global__ void CheckNonNegKernel(const size_t size, const T *input, T *out) {
  out[0] = 0;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (input[pos] < 0) {
      out[0] = 1;
    }
  }
  return;
}

template <typename T>
hipError_t CheckNonNeg(const size_t size, const T *input, T *output, hipStream_t hip_stream) {
  CheckNonNegKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return GetCudaStatus();
}

template <typename T>
__device__ int BinarySearchForMultinomial(T *start_addr, int size, T rand) {
  int start = 0;
  int end = size;
  while (end - start > 0) {
    int mid = start + (end - start) / 2;
    T mid_val = start_addr[mid];
    if (mid_val < rand) {
      start = mid + 1;
    } else {
      end = mid;
    }
  }
  if (start == size) {
    start = size - 1;
  }
  return start;
}

template <typename T, typename S>
__global__ void MultinomialKernel(int row, int col, T *probs, hiprandState *state, int64_t *num_sample, S *output) {
  // Load the probs to shared memory.
  extern __shared__ float accum_probs[];
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int probs_base_index = gid * col;
  if (probs_base_index >= row * col) {
    return;
  }

  int shm_base_index = threadIdx.x * col;
  accum_probs[shm_base_index] = probs[probs_base_index];
  for (int i = 1; i < col; i++) {
    probs_base_index++;
    float prob = static_cast<float>(probs[probs_base_index]);
    CUDA_KERNEL_ASSERT(prob >= 0);
    CUDA_KERNEL_ASSERT(!isnan(prob));
    CUDA_KERNEL_ASSERT(!isinf(prob));
    accum_probs[shm_base_index + i] = accum_probs[shm_base_index + i - 1] + prob;
  }
  __syncthreads();

  // Probs normalization.
  float max_probs = accum_probs[shm_base_index + col - 1];
  for (int i = 0; i < col; i++) {
    accum_probs[shm_base_index + i] /= max_probs;
  }
  __syncthreads();

  // Sample.
  int output_base_index = gid * num_sample[0];
  auto local_state = state[gid];
  for (int i = 0; i < num_sample[0]; i++) {
    float rand = hiprand_uniform(&local_state);
    output[output_base_index + i] = static_cast<S>(BinarySearchForMultinomial(&accum_probs[shm_base_index], col, rand));
  }
  state[gid] = local_state;
}

template <typename T, typename S>
hipError_t Multinomial(int row, int col, T *probs, hiprandState *state, int64_t *num_sample, S *output,
                        hipStream_t stream) {
  // Every block process several rows. It depends on shared memory usage.
  constexpr int max_shm_used_per_block = 256;
  int block_dim = std::max(Floor(std::min(row, max_shm_used_per_block), col), 1);
  int grid_dim = Ceil(row, block_dim);
  int shm_size = block_dim * col * sizeof(float);

  MultinomialKernel<<<grid_dim, block_dim, shm_size, stream>>>(row, col, probs, state, num_sample, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Multinomial<float, int64_t>(int row, int col, float *probs, hiprandState *state,
                                                                 int64_t *num_sample, int64_t *output,
                                                                 hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<double, int64_t>(int row, int col, double *probs, hiprandState *state,
                                                                  int64_t *num_sample, int64_t *output,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<half, int64_t>(int row, int col, half *probs, hiprandState *state,
                                                                int64_t *num_sample, int64_t *output,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int8_t, int64_t>(int row, int col, int8_t *probs, hiprandState *state,
                                                                  int64_t *num_sample, int64_t *output,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int16_t, int64_t>(int row, int col, int16_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int64_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int32_t, int64_t>(int row, int col, int32_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int64_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int64_t, int64_t>(int row, int col, int64_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int64_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint8_t, int64_t>(int row, int col, uint8_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int64_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint16_t, int64_t>(int row, int col, uint16_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint32_t, int64_t>(int row, int col, uint32_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint64_t, int64_t>(int row, int col, uint64_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<float, int32_t>(int row, int col, float *probs, hiprandState *state,
                                                                 int64_t *num_sample, int32_t *output,
                                                                 hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<double, int32_t>(int row, int col, double *probs, hiprandState *state,
                                                                  int64_t *num_sample, int32_t *output,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<half, int32_t>(int row, int col, half *probs, hiprandState *state,
                                                                int64_t *num_sample, int32_t *output,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int8_t, int32_t>(int row, int col, int8_t *probs, hiprandState *state,
                                                                  int64_t *num_sample, int32_t *output,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int16_t, int32_t>(int row, int col, int16_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int32_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int32_t, int32_t>(int row, int col, int32_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int32_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<int64_t, int32_t>(int row, int col, int64_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int32_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint8_t, int32_t>(int row, int col, uint8_t *probs, hiprandState *state,
                                                                   int64_t *num_sample, int32_t *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint16_t, int32_t>(int row, int col, uint16_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int32_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint32_t, int32_t>(int row, int col, uint32_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int32_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Multinomial<uint64_t, int32_t>(int row, int col, uint64_t *probs,
                                                                    hiprandState *state, int64_t *num_sample,
                                                                    int32_t *output, hipStream_t stream);

template CUDA_LIB_EXPORT hipError_t CheckNonNeg<float>(const size_t size, const float *input, float *output,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CheckZero<float>(const size_t distributions, const size_t categories,
                                                      const float *input, float *output, hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/tensor_scatter_elements.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
struct ReductionAssignment {
  __device__ void operator()(T *a, const T b) const { (*a) = b; }
};

template <typename T>
struct ReductionAdd {
  __device__ void operator()(T *a, const T b) const { return (void)MsAtomicAdd<T>(a, b); }
};

template <typename T, typename S, typename ReductionT>
__global__ void TensorScatterElementsKernel(const int input_dims, const int indices_size, const S *indices,
                                            const T *updates, T *output, const int64_t axis,
                                            const int64_t input_axis_size, const size_t *indices_stride,
                                            const size_t *output_stride, const ReductionT reduction_func) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < indices_size; index += blockDim.x * gridDim.x) {
    int remain = index;
    int output_offset = 0;
    for (size_t i = 0; i < input_dims; ++i) {
      int output_dim_index = remain / indices_stride[i];
      remain %= indices_stride[i];
      if (i == axis) {
        output_dim_index = *(indices + index);
        if (output_dim_index >= input_axis_size || output_dim_index < -input_axis_size) {
          return;
        }
        if (output_dim_index < 0) {
          output_dim_index += input_axis_size;
        }
      }
      output_offset += output_stride[i] * output_dim_index;
    }
    reduction_func(output + output_offset, *(updates + index));
  }
  return;
}

template <typename T, typename S>
hipError_t TensorScatterElements(const enum TensorScatterElementsReductionType reduction_type, const int input_dims,
                                  const int indices_size, const S *indices, const T *updates, T *output,
                                  const int64_t axis, const int64_t input_axis_size, const size_t *indices_stride,
                                  const size_t *output_stride, const uint32_t &device_id, hipStream_t stream) {
  switch (reduction_type) {
    case REDUCTION_ASSIGNMENT:
      TensorScatterElementsKernel<T, S><<<CUDA_BLOCKS(device_id, indices_size), CUDA_THREADS(device_id), 0, stream>>>(
        input_dims, indices_size, indices, updates, output, axis, input_axis_size, indices_stride, output_stride,
        ReductionAssignment<T>());
      break;
    case REDUCTION_ADD:
      TensorScatterElementsKernel<T, S><<<CUDA_BLOCKS(device_id, indices_size), CUDA_THREADS(device_id), 0, stream>>>(
        input_dims, indices_size, indices, updates, output, axis, input_axis_size, indices_stride, output_stride,
        ReductionAdd<T>());
      break;
    default:
      break;
  }
  return GetCudaStatus();
}

#define SCATTER_ELEMENTS_FUNC(T, S)                                                                             \
  template CUDA_LIB_EXPORT hipError_t TensorScatterElements(                                                   \
    const enum TensorScatterElementsReductionType reduction_type, const int input_dims, const int indices_size, \
    const S *indices, const T *updates, T *output, const int64_t axis, const int64_t input_axis_size,           \
    const size_t *indices_stride, const size_t *output_stride, const uint32_t &device_id, hipStream_t stream)

#define SCATTER_ELEMENTS_INDEX_FUNC(T) \
  SCATTER_ELEMENTS_FUNC(T, int32_t);   \
  SCATTER_ELEMENTS_FUNC(T, int64_t);

SCATTER_ELEMENTS_INDEX_FUNC(half)
SCATTER_ELEMENTS_INDEX_FUNC(float)
SCATTER_ELEMENTS_INDEX_FUNC(double)
SCATTER_ELEMENTS_INDEX_FUNC(int8_t)
SCATTER_ELEMENTS_INDEX_FUNC(uint8_t)
SCATTER_ELEMENTS_INDEX_FUNC(int16_t)
SCATTER_ELEMENTS_INDEX_FUNC(int)
SCATTER_ELEMENTS_INDEX_FUNC(bool)
SCATTER_ELEMENTS_INDEX_FUNC(uint32_t)
SCATTER_ELEMENTS_INDEX_FUNC(uint16_t)
SCATTER_ELEMENTS_INDEX_FUNC(int64_t)
SCATTER_ELEMENTS_INDEX_FUNC(uint64_t)

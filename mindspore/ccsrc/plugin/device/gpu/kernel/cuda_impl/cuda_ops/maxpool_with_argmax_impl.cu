#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "maxpool_with_argmax_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T, typename S>
__global__ void MaxPoolWithArgmax(const T *input, const int n, const int c, const int h, const int w,
                                  const int windowHeight, const int windowWidth, const int strideHeight,
                                  const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
                                  const int outputWidth, const int outputNCHW, const int outputCHW, const int outputHW,
                                  T *output, S *index) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (outputNCHW); pos += blockDim.x * gridDim.x) {
    const int posn = pos / outputCHW;
    const int posc = pos / outputHW % c;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;
    int hstart = posh * strideHeight - padTop;
    int wstart = posw * strideWidth - padLeft;
    const int hend = min(hstart + windowHeight, h);
    const int wend = min(wstart + windowWidth, w);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    S inputStart = posn * c * h * w;
    S maxIdx = posc * h * w + hstart * w + wstart;
    T maxData = input[inputStart + maxIdx];
    for (int hcur = hstart; hcur < hend; ++hcur) {
      for (int wcur = wstart; wcur < wend; ++wcur) {
        S inputIdx = posc * h * w + hcur * w + wcur;
        T inputData = input[inputStart + inputIdx];
        if (inputData > maxData) {
          maxIdx = inputIdx;
          maxData = inputData;
        }
      }
    }
    output[pos] = maxData;
    index[pos] = maxIdx;
  }
}

template <typename T, typename S>
hipError_t CalMaxPoolWithArgmax(const T *input, const int n, const int c, const int h, const int w,
                                 const int windowHeight, const int windowWidth, const int strideHeight,
                                 const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
                                 const int outputWidth, T *output, S *index, const uint32_t &device_id,
                                 hipStream_t hip_stream) {
  const int outputNCHW = n * c * outputHeight * outputWidth;
  const int outputCHW = c * outputHeight * outputWidth;
  const int outputHW = outputHeight * outputWidth;
  MaxPoolWithArgmax<<<CUDA_BLOCKS(device_id, n * c * outputHeight * outputWidth), CUDA_THREADS(device_id), 0,
                      hip_stream>>>(input, n, c, h, w, windowHeight, windowWidth, strideHeight, strideWidth, padTop,
                                     padLeft, outputHeight, outputWidth, outputNCHW, outputCHW, outputHW, output,
                                     index);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<int8_t, int>(
  const int8_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, int8_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<int16_t, int>(
  const int16_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, int16_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<int64_t, int>(
  const int64_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, int64_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<uint8_t, int>(
  const uint8_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, uint8_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<uint16_t, int>(
  const uint16_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, uint16_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<uint32_t, int>(
  const uint32_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, uint32_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<uint64_t, int>(
  const uint64_t *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, uint64_t *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<half, int>(
  const half *input, const int n, const int c, const int h, const int w, const int windowHeight, const int windowWidth,
  const int strideHeight, const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
  const int outputWidth, half *output, int *index, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<float, int>(
  const float *input, const int n, const int c, const int h, const int w, const int windowHeight, const int windowWidth,
  const int strideHeight, const int strideWidth, const int padTop, const int padLeft, const int outputHeight,
  const int outputWidth, float *output, int *index, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxPoolWithArgmax<double, int>(
  const double *input, const int n, const int c, const int h, const int w, const int windowHeight,
  const int windowWidth, const int strideHeight, const int strideWidth, const int padTop, const int padLeft,
  const int outputHeight, const int outputWidth, double *output, int *index, const uint32_t &device_id,
  hipStream_t hip_stream);

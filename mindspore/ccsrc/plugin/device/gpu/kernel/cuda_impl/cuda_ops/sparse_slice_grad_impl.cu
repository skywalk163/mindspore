#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/sparse_slice_grad_impl.cuh"
#include <algorithm>
#include <complex>
#include "plugin/device/cpu/kernel/nnacl/op_base.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"

template <typename DataType, typename IndexType>
__global__ void SparseSliceGradKernel(const DataType *x_ptr, const IndexType *indices_ptr, const IndexType *start_ptr,
                                      const IndexType *new_indices_ptr, DataType *y_ptr, size_t *num_propagated,
                                      size_t input_nnz_, size_t output_nnz_, size_t num_dims_) {
  size_t input_nz = blockIdx.x * blockDim.x + threadIdx.x;
  if (input_nz < input_nnz_) {
    y_ptr[input_nz] = DataType(0);
    for (IndexType output_nz = 0; output_nz < output_nnz_; ++output_nz) {
      bool is_same = true;
      for (int d = 0; d < num_dims_; ++d) {
        const IndexType a = indices_ptr[input_nz * num_dims_ + d];
        const IndexType b = new_indices_ptr[output_nz * num_dims_ + d];
        const IndexType offset = start_ptr[d];
        if (b + offset != a) {
          is_same = false;
          break;
        }
      }
      if (is_same) {
        y_ptr[input_nz] = x_ptr[output_nz];
        MsAtomicAdd(num_propagated, size_t(1));
        break;
      }
    }
  }
}

template <typename DataType, typename IndexType>
CUDA_LIB_EXPORT hipError_t SparseSliceGrad(const DataType *x_ptr, const IndexType *indices_ptr,
                                            const IndexType *start_ptr, const IndexType *new_indices_ptr,
                                            DataType *y_ptr, size_t *num_propagated, size_t input_nnz_,
                                            size_t output_nnz_, size_t num_dim_, uint32_t device_id,
                                            hipStream_t hip_stream) {
  int threads_per_block = CUDA_THREADS(device_id);
  unsigned int grid_num = UP_DIV(input_nnz_ + 1, threads_per_block);
  SparseSliceGradKernel<<<grid_num, threads_per_block, 0, hip_stream>>>(
    x_ptr, indices_ptr, start_ptr, new_indices_ptr, y_ptr, num_propagated, input_nnz_, output_nnz_, num_dim_);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<int8_t, int64_t>(const int8_t *x_ptr, const int64_t *indices_ptr,
                                                                      const int64_t *start_ptr,
                                                                      const int64_t *new_indices_ptr, int8_t *y_ptr,
                                                                      size_t *num_propagated, size_t input_nnz_,
                                                                      size_t output_nnz_, size_t num_dim_,
                                                                      uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<int16_t, int64_t>(const int16_t *x_ptr, const int64_t *indices_ptr,
                                                                       const int64_t *start_ptr,
                                                                       const int64_t *new_indices_ptr, int16_t *y_ptr,
                                                                       size_t *num_propagated, size_t input_nnz_,
                                                                       size_t output_nnz_, size_t num_dim_,
                                                                       uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<int32_t, int64_t>(const int32_t *x_ptr, const int64_t *indices_ptr,
                                                                       const int64_t *start_ptr,
                                                                       const int64_t *new_indices_ptr, int32_t *y_ptr,
                                                                       size_t *num_propagated, size_t input_nnz_,
                                                                       size_t output_nnz_, size_t num_dim_,
                                                                       uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<int64_t, int64_t>(const int64_t *x_ptr, const int64_t *indices_ptr,
                                                                       const int64_t *start_ptr,
                                                                       const int64_t *new_indices_ptr, int64_t *y_ptr,
                                                                       size_t *num_propagated, size_t input_nnz_,
                                                                       size_t output_nnz_, size_t num_dim_,
                                                                       uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<uint8_t, int64_t>(const uint8_t *x_ptr, const int64_t *indices_ptr,
                                                                       const int64_t *start_ptr,
                                                                       const int64_t *new_indices_ptr, uint8_t *y_ptr,
                                                                       size_t *num_propagated, size_t input_nnz_,
                                                                       size_t output_nnz_, size_t num_dim_,
                                                                       uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<uint16_t, int64_t>(
  const uint16_t *x_ptr, const int64_t *indices_ptr, const int64_t *start_ptr, const int64_t *new_indices_ptr,
  uint16_t *y_ptr, size_t *num_propagated, size_t input_nnz_, size_t output_nnz_, size_t num_dim_, uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<uint32_t, int64_t>(
  const uint32_t *x_ptr, const int64_t *indices_ptr, const int64_t *start_ptr, const int64_t *new_indices_ptr,
  uint32_t *y_ptr, size_t *num_propagated, size_t input_nnz_, size_t output_nnz_, size_t num_dim_, uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<uint64_t, int64_t>(
  const uint64_t *x_ptr, const int64_t *indices_ptr, const int64_t *start_ptr, const int64_t *new_indices_ptr,
  uint64_t *y_ptr, size_t *num_propagated, size_t input_nnz_, size_t output_nnz_, size_t num_dim_, uint32_t device_id,
  hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<half, int64_t>(const half *x_ptr, const int64_t *indices_ptr,
                                                                    const int64_t *start_ptr,
                                                                    const int64_t *new_indices_ptr, half *y_ptr,
                                                                    size_t *num_propagated, size_t input_nnz_,
                                                                    size_t output_nnz_, size_t num_dim_,
                                                                    uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<float, int64_t>(const float *x_ptr, const int64_t *indices_ptr,
                                                                     const int64_t *start_ptr,
                                                                     const int64_t *new_indices_ptr, float *y_ptr,
                                                                     size_t *num_propagated, size_t input_nnz_,
                                                                     size_t output_nnz_, size_t num_dim_,
                                                                     uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<double, int64_t>(const double *x_ptr, const int64_t *indices_ptr,
                                                                      const int64_t *start_ptr,
                                                                      const int64_t *new_indices_ptr, double *y_ptr,
                                                                      size_t *num_propagated, size_t input_nnz_,
                                                                      size_t output_nnz_, size_t num_dim_,
                                                                      uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<bool, int64_t>(const bool *x_ptr, const int64_t *indices_ptr,
                                                                    const int64_t *start_ptr,
                                                                    const int64_t *new_indices_ptr, bool *y_ptr,
                                                                    size_t *num_propagated, size_t input_nnz_,
                                                                    size_t output_nnz_, size_t num_dim_,
                                                                    uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<Complex<float>, int64_t>(
  const Complex<float> *x_ptr, const int64_t *indices_ptr, const int64_t *start_ptr, const int64_t *new_indices_ptr,
  Complex<float> *y_ptr, size_t *num_propagated, size_t input_nnz_, size_t output_nnz_, size_t num_dim_,
  uint32_t device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SparseSliceGrad<Complex<double>, int64_t>(
  const Complex<double> *x_ptr, const int64_t *indices_ptr, const int64_t *start_ptr, const int64_t *new_indices_ptr,
  Complex<double> *y_ptr, size_t *num_propagated, size_t input_nnz_, size_t output_nnz_, size_t num_dim_,
  uint32_t device_id, hipStream_t hip_stream);

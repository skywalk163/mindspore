#include "hip/hip_runtime.h"
/**
 * Copyright 2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_ops_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_common.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/binary_pub_impl.cuh"

template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<BinaryOpType::kMul, In0_t, In1_t, Out_t> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __forceinline__ Out_t operator()(In0_t val0, In1_t val1) const { return val0 * val1; }
};

template <>
struct BinaryFunc<BinaryOpType::kMul, bool, bool, bool> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __forceinline__ bool operator()(bool val0, bool val1) const { return val0 && val1; }
};
REGISTER_BINARY_OP_CUDA_FUNC_INT_TYPE(BinaryOpType::kMul);
REGISTER_BINARY_OP_CUDA_FUNC_FLOAT_TYPE(BinaryOpType::kMul);
REGISTER_BINARY_OP_CUDA_FUNC_COMPLEX_TYPE(BinaryOpType::kMul);
REGISTER_BINARY_OP_CUDA_FUNC_BOOL_TYPE(BinaryOpType::kMul);

// MulNoNan
template <typename T>
struct BinaryFunc<BinaryOpType::kMulNoNan, T, T, T, typename std::is_floating_point<T>::type> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ T operator()(const T &lhs, const T &rhs) const {
    return rhs < Eps<T>() && rhs > -Eps<T>() ? 0.0 : (lhs * rhs);
  }
};
template <typename T>
struct BinaryFunc<BinaryOpType::kMulNoNan, T, T, T, typename std::is_integral<T>::type> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ T operator()(const T &lhs, const T &rhs) const {
    return rhs == 0 ? 0 : (lhs * rhs);
  }
};
template <>
struct BinaryFunc<BinaryOpType::kMulNoNan, half, half, half> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ half operator()(const half &lhs, const half &rhs) const {
    bool bool1 = __half2float(rhs) < (0.00001) && __half2float(rhs) > -0.00001;
    if (bool1) {
      return static_cast<half>(0.0);
    }
    return __float2half_rn(__half2float(lhs) * __half2float(rhs));
  }
};

template <typename In0_t, typename In1_t, typename Out_t>
struct BinaryFunc<BinaryOpType::kMulNoNan, In0_t, In1_t, Complex<Out_t>> {
  __device__ __host__ __forceinline__ BinaryFunc() {}
  __device__ __host__ __forceinline__ Complex<Out_t> operator()(const In0_t &lhs, const In1_t &rhs) const {
    Complex<Out_t> complex_rhs(rhs);
    if ((complex_rhs.real() < Eps<float>() && complex_rhs.real() > -Eps<float>()) ||
        (complex_rhs.imag() < Eps<float>() && complex_rhs.imag() > -Eps<float>())) {
      Complex<Out_t> res(0.0, 0.0);
      return res;
    }
    return lhs * rhs;
  }
};
REGISTER_BINARY_OP_CUDA_FUNC_INT_TYPE(BinaryOpType::kMulNoNan);
REGISTER_BINARY_OP_CUDA_FUNC_COMPLEX_TYPE(BinaryOpType::kMulNoNan);
REGISTER_BINARY_OP_CUDA_FUNC_FLOAT_TYPE(BinaryOpType::kMulNoNan);

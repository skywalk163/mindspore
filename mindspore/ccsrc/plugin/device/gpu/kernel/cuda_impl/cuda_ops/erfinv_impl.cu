#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/erfinv_impl.cuh"
#include "include/hip/hip_fp16.h"
#include "plugin/device/cpu/kernel/nnacl/op_base.h"

constexpr uint elements_per_thread = 4;
constexpr uint threads_per_block = 256;
constexpr uint elements_per_block = elements_per_thread * threads_per_block;

template <typename T>
struct VectorizedTrait {  // Only use of raw pointer with no offset.
  static const uint VecSize = 4;
};

template <>
struct VectorizedTrait<half> {
  static const uint VecSize = 2;
};

template <typename T, int VecSize>
struct alignas(sizeof(T) * VecSize) AlignVec {
  T data[VecSize];
};

template <typename Func, typename T>
__device__ __forceinline__ void VectorizedCall(Func func, const T *in, T *out) {
  constexpr uint vec_size = VectorizedTrait<T>::VecSize;
  constexpr uint elements_per_loop = elements_per_thread / vec_size;
  using VecT = AlignVec<T, vec_size>;

  uint tid = threadIdx.x;
  auto vec_in = reinterpret_cast<const VecT *>(in);
  auto vec_out = reinterpret_cast<VecT *>(out);

#pragma unroll
  for (uint i = 0; i < elements_per_loop; i++) {
    uint index = tid + i * threads_per_block;
    VecT cache = vec_in[index];
#pragma unroll
    for (uint j = 0; j < vec_size; j++) {
      cache.data[j] = func(cache.data[j]);
    }
    vec_out[index] = cache;
  }
}

template <typename Func, typename T>
__device__ __forceinline__ void NormalCall(Func func, const T *in, T *out, uint remaining) {
  uint loop = UP_DIV(remaining, elements_per_thread);
  for (uint i = threadIdx.x; i < loop; i += blockDim.x) {
#pragma unroll
    for (uint j = 0; j < elements_per_thread; j++) {
      uint index = i * elements_per_thread + j;
      if (index >= remaining) {
        return;
      }
      out[index] = func(in[index]);
    }
  }
}

template <typename Func, typename T>
__global__ void VectorizedFor(Func func, const T *in, T *out, uint num_of_elements) {
  uint offset = elements_per_block * blockIdx.x;
  uint remaining = num_of_elements - offset;

  if (blockIdx.x + 1 == gridDim.x && remaining != elements_per_block) {
    NormalCall(func, in + offset, out + offset, remaining);
  } else {
    VectorizedCall(func, in + offset, out + offset);
  }
}

template <typename T>
struct ErfinvFunctor {
  __device__ __forceinline__ T operator()(T x) const { return erfinv(x); }
};

template <>
struct ErfinvFunctor<half> {
  __device__ __forceinline__ half operator()(half x) const { return __float2half(erfinvf(__half2float(x))); }
};

template <>
struct ErfinvFunctor<double> {
  __device__ __forceinline__ double operator()(double x) const { return erfinvf(x); }
};

template <typename T>
hipError_t Erfinv(size_t input_size, const T *input, T *output, const uint32_t &device_id, hipStream_t hip_stream) {
  ErfinvFunctor<T> functor{};
  auto block_x = threads_per_block;
  auto grid_x = UP_DIV(static_cast<uint>(input_size), elements_per_block);
  dim3 block{block_x};
  dim3 grid{grid_x};
  VectorizedFor<<<grid, block, 0, hip_stream>>>(functor, input, output, static_cast<uint>(input_size));
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Erfinv<float>(size_t input_size, const float *input, float *output,
                                                   const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t Erfinv<half>(size_t input_size, const half *input, half *output,
                                                  const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t Erfinv<double>(size_t input_size, const double *input, double *output,
                                                    const uint32_t &device_id, hipStream_t hip_stream);

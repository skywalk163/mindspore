#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/dilation2d_backprop_filter_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void InitOutput(T *output, const int64_t outer_size) {
  T zero = 0;
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < outer_size; id += blockDim.x * gridDim.x) {
    output[id] = zero;
  }
  return;
}

template <typename T>
__global__ void Dilation2DBackpropFilter(const T *input, const T *filter, const T *out_backprop, T *output,
                                         const int64_t inputHeight, const int64_t inputWidth, const int64_t Channel,
                                         const int64_t filterHeight, const int64_t filterWidth,
                                         const int64_t outputHeight, const int64_t outputWidth,
                                         const int64_t strideHeight, const int64_t strideWidth,
                                         const int64_t rateHeight, const int64_t rateWidth, const int64_t pad_top,
                                         const int64_t pad_left, const int64_t outer_size) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < outer_size; pos += blockDim.x * gridDim.x) {
    const int posn = pos / (outputHeight * outputWidth * Channel);
    const int posc = pos / (outputHeight * outputWidth) % Channel;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;
    const int height_start = posh * strideHeight - pad_top;
    const int width_start = posw * strideWidth - pad_left;

    T max_val = std::numeric_limits<T>::lowest();
    int max_h = 0;
    int max_w = 0;

    for (int h = 0; h < filterHeight; ++h) {
      const int h_in = height_start + h * rateHeight;
      if (h_in >= 0 && h_in < inputHeight) {
        for (int w = 0; w < filterWidth; ++w) {
          const int w_in = width_start + w * rateWidth;
          if (w_in >= 0 && w_in < inputWidth) {
            const T val = input[w_in + inputWidth * (h_in + inputHeight * (posc + Channel * posn))] +
                          filter[w + filterWidth * (h + filterHeight * posc)];
            if (val > max_val) {
              max_val = val;
              max_h = h;
              max_w = w;
            }
          }
        }
      }
    }
    MsAtomicAdd(output + max_w + filterWidth * (max_h + filterHeight * posc), out_backprop[pos]);
  }
  return;
}

template <>
__global__ void Dilation2DBackpropFilter(const half *input, const half *filter, const half *out_backprop, half *output,
                                         const int64_t inputHeight, const int64_t inputWidth, const int64_t Channel,
                                         const int64_t filterHeight, const int64_t filterWidth,
                                         const int64_t outputHeight, const int64_t outputWidth,
                                         const int64_t strideHeight, const int64_t strideWidth,
                                         const int64_t rateHeight, const int64_t rateWidth, const int64_t pad_top,
                                         const int64_t pad_left, const int64_t outer_size) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < outer_size; pos += blockDim.x * gridDim.x) {
    const int posn = pos / (outputHeight * outputWidth * Channel);
    const int posc = pos / (outputHeight * outputWidth) % Channel;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;
    const int height_start = posh * strideHeight - pad_top;
    const int width_start = posw * strideWidth - pad_left;

    half max_val = __int2half_rd(-65504);
    int max_h = 0;
    int max_w = 0;

    for (int h = 0; h < filterHeight; ++h) {
      const int h_in = height_start + h * rateHeight;
      if (h_in >= 0 && h_in < inputHeight) {
        for (int w = 0; w < filterWidth; ++w) {
          const int w_in = width_start + w * rateWidth;
          if (w_in >= 0 && w_in < inputWidth) {
            const half val = input[w_in + inputWidth * (h_in + inputHeight * (posc + Channel * posn))] +
                             filter[w + filterWidth * (h + filterHeight * posc)];
            if (val > max_val) {
              max_val = val;
              max_h = h;
              max_w = w;
            }
          }
        }
      }
    }
    MsAtomicAdd(output + max_w + filterWidth * (max_h + filterHeight * posc), out_backprop[pos]);
  }
  return;
}

template <typename T>
hipError_t CalDilation2DBackpropFilter(const T *input, const T *filter, const T *out_backprop, T *output,
                                        const std::vector<int64_t> &input_shape,
                                        const std::vector<int64_t> &filter_shape,
                                        const std::vector<int64_t> &out_backprop_shape,
                                        const std::vector<int64_t> &output_shape, const std::vector<int64_t> &stride,
                                        const std::vector<int64_t> &dilation, int64_t (&pads)[2],
                                        const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream) {
  InitOutput<<<CUDA_BLOCKS(device_id, output_shape[0] * output_shape[1] * output_shape[2]), CUDA_THREADS(device_id), 0,
               hip_stream>>>(output, output_shape[0] * output_shape[1] * output_shape[2]);
  Dilation2DBackpropFilter<<<CUDA_BLOCKS(device_id, outer_size), CUDA_THREADS(device_id), 0, hip_stream>>>(
    input, filter, out_backprop, output, input_shape[2], input_shape[3], input_shape[1], filter_shape[1],
    filter_shape[2], out_backprop_shape[2], out_backprop_shape[3], stride[2], stride[3], dilation[2], dilation[3],
    pads[0], pads[1], outer_size);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<half>(
  const half *input, const half *filter, const half *out_backprop, half *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<float>(
  const float *input, const float *filter, const float *out_backprop, float *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<double>(
  const double *input, const double *filter, const double *out_backprop, double *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<int32_t>(
  const int32_t *input, const int32_t *filter, const int32_t *out_backprop, int32_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<int64_t>(
  const int64_t *input, const int64_t *filter, const int64_t *out_backprop, int64_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<int8_t>(
  const int8_t *input, const int8_t *filter, const int8_t *out_backprop, int8_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<int16_t>(
  const int16_t *input, const int16_t *filter, const int16_t *out_backprop, int16_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<uint8_t>(
  const uint8_t *input, const uint8_t *filter, const uint8_t *out_backprop, uint8_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<uint16_t>(
  const uint16_t *input, const uint16_t *filter, const uint16_t *out_backprop, uint16_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<uint32_t>(
  const uint32_t *input, const uint32_t *filter, const uint32_t *out_backprop, uint32_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t CalDilation2DBackpropFilter<uint64_t>(
  const uint64_t *input, const uint64_t *filter, const uint64_t *out_backprop, uint64_t *output,
  const std::vector<int64_t> &input_shape, const std::vector<int64_t> &filter_shape,
  const std::vector<int64_t> &out_backprop_shape, const std::vector<int64_t> &output_shape,
  const std::vector<int64_t> &stride, const std::vector<int64_t> &dilation, int64_t (&pads)[2],
  const int64_t outer_size, const uint32_t &device_id, hipStream_t hip_stream);

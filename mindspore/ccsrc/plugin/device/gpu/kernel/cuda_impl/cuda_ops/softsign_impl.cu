#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/softsign_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>

__global__ void SoftsignKernel(const size_t size, const T *input_addr, T *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = input_addr[pos] / (1. + abs(input_addr[pos]));
  }
}

template <>
__global__ void SoftsignKernel(const size_t size, const half *input_addr, half *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = __half2float(input_addr[pos]) / (1. + abs(__half2float(input_addr[pos])));
  }
}

template <>
__global__ void SoftsignKernel(const size_t size, const double *input_addr, double *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = input_addr[pos] / (1. + abs(input_addr[pos]));
  }
}

template <typename T>
hipError_t Softsign(const size_t size, const T *input_addr, T *output_addr, const uint32_t &device_id,
                     hipStream_t hip_stream) {
  SoftsignKernel<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input_addr,
                                                                                            output_addr);
  return GetCudaStatus();
}

template <>
hipError_t Softsign(const size_t size, const half *input_addr, half *output_addr, const uint32_t &device_id,
                     hipStream_t hip_stream) {
  SoftsignKernel<half>
    <<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input_addr, output_addr);
  return GetCudaStatus();
}

template <>
hipError_t Softsign(const size_t size, const double *input_addr, double *output_addr, const uint32_t &device_id,
                     hipStream_t hip_stream) {
  SoftsignKernel<double>
    <<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input_addr, output_addr);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Softsign<float>(const size_t size, const float *input_addr, float *output_addr,
                                                     const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t Softsign<half>(const size_t size, const half *input_addr, half *output_addr,
                                                    const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t Softsign<double>(const size_t size, const double *input_addr, double *output_addr,
                                                      const uint32_t &device_id, hipStream_t hip_stream);

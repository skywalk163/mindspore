#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/scatter_nd.cuh"
#include <complex>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void ScatterNdKernel(S *indices, T *update, T *output, const size_t block_size, const size_t input_size,
                                const size_t output_size, const size_t indices_dim_0, const size_t indices_dim_1,
                                const ScatterNdInfo<S> info) {
  const S *indices_stride = info.indices_stride;
  const S *work_shape = info.shape;
  int i, j;
  for (size_t read_index = blockIdx.x * blockDim.x + threadIdx.x; read_index < input_size;
       read_index += blockDim.x * gridDim.x) {
    size_t write_index = 0;
    bool out_bound = false;

    i = read_index / block_size;
    j = read_index % block_size;

    for (size_t k = 0; k < indices_dim_1; k++) {
      S indices_i = indices[i * indices_dim_1 + k];
      CUDA_KERNEL_ASSERT(indices_i >= 0 && indices_i < work_shape[k]);
      write_index += indices_i * indices_stride[k];
    }

    write_index += j;
    CUDA_KERNEL_ASSERT(write_index < output_size);

    MsAtomicAdd(&output[write_index], update[read_index]);
  }
}

template <typename T, typename S>
hipError_t ScatterNd(S *indices, T *update, T *output, const size_t &block_size, const size_t &input_size,
                      const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
                      const ScatterNdInfo<S> &info, hipStream_t stream) {
  ScatterNdKernel<<<GET_BLOCKS(output_size), GET_THREADS, 0, stream>>>(indices, update, output, block_size, input_size,
                                                                       output_size, indices_dim_0, indices_dim_1, info);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t ScatterNd<double, int16_t>(int16_t *indices, double *update, double *output,
                                                                const size_t &block_size, const size_t &input_size,
                                                                const size_t &output_size, const size_t &indices_dim_0,
                                                                const size_t &indices_dim_1,
                                                                const ScatterNdInfo<int16_t> &info,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<double, int>(int *indices, double *update, double *output,
                                                            const size_t &block_size, const size_t &input_size,
                                                            const size_t &output_size, const size_t &indices_dim_0,
                                                            const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                            hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<double, int64_t>(int64_t *indices, double *update, double *output,
                                                                const size_t &block_size, const size_t &input_size,
                                                                const size_t &output_size, const size_t &indices_dim_0,
                                                                const size_t &indices_dim_1,
                                                                const ScatterNdInfo<int64_t> &info,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<float, int16_t>(int16_t *indices, float *update, float *output,
                                                               const size_t &block_size, const size_t &input_size,
                                                               const size_t &output_size, const size_t &indices_dim_0,
                                                               const size_t &indices_dim_1,
                                                               const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<float, int>(int *indices, float *update, float *output,
                                                           const size_t &block_size, const size_t &input_size,
                                                           const size_t &output_size, const size_t &indices_dim_0,
                                                           const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                           hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<float, int64_t>(int64_t *indices, float *update, float *output,
                                                               const size_t &block_size, const size_t &input_size,
                                                               const size_t &output_size, const size_t &indices_dim_0,
                                                               const size_t &indices_dim_1,
                                                               const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<half, int16_t>(int16_t *indices, half *update, half *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<half, int>(int *indices, half *update, half *output,
                                                          const size_t &block_size, const size_t &input_size,
                                                          const size_t &output_size, const size_t &indices_dim_0,
                                                          const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                          hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<half, int64_t>(int64_t *indices, half *update, half *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int64_t, int16_t>(int16_t *indices, int64_t *update, int64_t *output,
                                                                 const size_t &block_size, const size_t &input_size,
                                                                 const size_t &output_size, const size_t &indices_dim_0,
                                                                 const size_t &indices_dim_1,
                                                                 const ScatterNdInfo<int16_t> &info,
                                                                 hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int64_t, int>(int *indices, int64_t *update, int64_t *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1,
                                                             const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int64_t, int64_t>(int64_t *indices, int64_t *update, int64_t *output,
                                                                 const size_t &block_size, const size_t &input_size,
                                                                 const size_t &output_size, const size_t &indices_dim_0,
                                                                 const size_t &indices_dim_1,
                                                                 const ScatterNdInfo<int64_t> &info,
                                                                 hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int, int16_t>(int16_t *indices, int *update, int *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1,
                                                             const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int, int>(int *indices, int *update, int *output,
                                                         const size_t &block_size, const size_t &input_size,
                                                         const size_t &output_size, const size_t &indices_dim_0,
                                                         const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                         hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int, int64_t>(int64_t *indices, int *update, int *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1,
                                                             const ScatterNdInfo<int64_t> &info, hipStream_t stream);
// NOLINTNEXTLINE
template CUDA_LIB_EXPORT hipError_t ScatterNd<short, int16_t>(int16_t *indices, short *update, short *output,
                                                               const size_t &block_size, const size_t &input_size,
                                                               const size_t &output_size, const size_t &indices_dim_0,
                                                               const size_t &indices_dim_1,
                                                               const ScatterNdInfo<int16_t> &info, hipStream_t stream);
// NOLINTNEXTLINE
template CUDA_LIB_EXPORT hipError_t ScatterNd<short, int>(int *indices, short *update, short *output,
                                                           const size_t &block_size, const size_t &input_size,
                                                           const size_t &output_size, const size_t &indices_dim_0,
                                                           const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                           hipStream_t stream);
// NOLINTNEXTLINE
template CUDA_LIB_EXPORT hipError_t ScatterNd<short, int64_t>(int64_t *indices, short *update, short *output,
                                                               const size_t &block_size, const size_t &input_size,
                                                               const size_t &output_size, const size_t &indices_dim_0,
                                                               const size_t &indices_dim_1,
                                                               const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int8_t, int16_t>(int16_t *indices, int8_t *update, int8_t *output,
                                                                const size_t &block_size, const size_t &input_size,
                                                                const size_t &output_size, const size_t &indices_dim_0,
                                                                const size_t &indices_dim_1,
                                                                const ScatterNdInfo<int16_t> &info,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int8_t, int>(int *indices, int8_t *update, int8_t *output,
                                                            const size_t &block_size, const size_t &input_size,
                                                            const size_t &output_size, const size_t &indices_dim_0,
                                                            const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                            hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<int8_t, int64_t>(int64_t *indices, int8_t *update, int8_t *output,
                                                                const size_t &block_size, const size_t &input_size,
                                                                const size_t &output_size, const size_t &indices_dim_0,
                                                                const size_t &indices_dim_1,
                                                                const ScatterNdInfo<int64_t> &info,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<unsigned char, int16_t>(
  int16_t *indices, unsigned char *update, unsigned char *output, const size_t &block_size, const size_t &input_size,
  const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<unsigned char, int>(int *indices, unsigned char *update,
                                                                   unsigned char *output, const size_t &block_size,
                                                                   const size_t &input_size, const size_t &output_size,
                                                                   const size_t &indices_dim_0,
                                                                   const size_t &indices_dim_1,
                                                                   const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<unsigned char, int64_t>(
  int64_t *indices, unsigned char *update, unsigned char *output, const size_t &block_size, const size_t &input_size,
  const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint16_t, int16_t>(int16_t *indices, uint16_t *update, uint16_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<uint16_t, int>(int *indices, uint16_t *update, uint16_t *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint16_t, int64_t>(int64_t *indices, uint16_t *update, uint16_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint32_t, int16_t>(int16_t *indices, uint32_t *update, uint32_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<uint32_t, int>(int *indices, uint32_t *update, uint32_t *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint32_t, int64_t>(int64_t *indices, uint32_t *update, uint32_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint64_t, int16_t>(int16_t *indices, uint64_t *update, uint64_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<uint64_t, int>(int *indices, uint64_t *update, uint64_t *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<uint64_t, int64_t>(int64_t *indices, uint64_t *update, uint64_t *output, const size_t &block_size,
                             const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                             const size_t &indices_dim_1, const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<bool, int16_t>(int16_t *indices, bool *update, bool *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<bool, int>(int *indices, bool *update, bool *output,
                                                          const size_t &block_size, const size_t &input_size,
                                                          const size_t &output_size, const size_t &indices_dim_0,
                                                          const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
                                                          hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<bool, int64_t>(int64_t *indices, bool *update, bool *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1,
                                                              const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<Complex<float>, int16_t>(
  int16_t *indices, Complex<float> *update, Complex<float> *output, const size_t &block_size, const size_t &input_size,
  const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t
ScatterNd<Complex<float>, int>(int *indices, Complex<float> *update, Complex<float> *output, const size_t &block_size,
                               const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0,
                               const size_t &indices_dim_1, const ScatterNdInfo<int> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<Complex<float>, int64_t>(
  int64_t *indices, Complex<float> *update, Complex<float> *output, const size_t &block_size, const size_t &input_size,
  const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int64_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<Complex<double>, int16_t>(
  int16_t *indices, Complex<double> *update, Complex<double> *output, const size_t &block_size,
  const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int16_t> &info, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<Complex<double>, int>(
  int *indices, Complex<double> *update, Complex<double> *output, const size_t &block_size, const size_t &input_size,
  const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1, const ScatterNdInfo<int> &info,
  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t ScatterNd<Complex<double>, int64_t>(
  int64_t *indices, Complex<double> *update, Complex<double> *output, const size_t &block_size,
  const size_t &input_size, const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
  const ScatterNdInfo<int64_t> &info, hipStream_t stream);

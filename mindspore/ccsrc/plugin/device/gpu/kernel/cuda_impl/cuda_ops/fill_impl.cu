#include "hip/hip_runtime.h"
/**
 * Copyright 2021-2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/fill_impl.cuh"
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"

template <typename T>
using Complex = mindspore::utils::Complex<T>;

template <typename T>
__global__ void FillKernel(const size_t m, const size_t n, const T *input, T *output) {
  size_t element_num = m * n;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < element_num; pos += blockDim.x * gridDim.x) {
    output[pos] = input[pos % n];
  }
}

template <typename T>
hipError_t Fill(const size_t &m, const size_t &n, const T *input, T *output, hipStream_t stream) {
  FillKernel<<<(m * n + 255) / 256, 256, 0, stream>>>(m, n, input, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Fill<float>(const size_t &m, const size_t &n, const float *input, float *output,
                                                 hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<half>(const size_t &m, const size_t &n, const half *input, half *output,
                                                hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<double>(const size_t &m, const size_t &n, const double *input, double *output,
                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<int8_t>(const size_t &m, const size_t &n, const int8_t *input, int8_t *output,
                                                  hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<Complex<float>>(const size_t &m, const size_t &n, const Complex<float> *input,
                                                          Complex<float> *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<Complex<double>>(const size_t &m, const size_t &n,
                                                           const Complex<double> *input, Complex<double> *output,
                                                           hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<int16_t>(const size_t &m, const size_t &n, const int16_t *input,
                                                   int16_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<int32_t>(const size_t &m, const size_t &n, const int32_t *input,
                                                   int32_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<int64_t>(const size_t &m, const size_t &n, const int64_t *input,
                                                   int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<uint16_t>(const size_t &m, const size_t &n, const uint16_t *input,
                                                    uint16_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<uint32_t>(const size_t &m, const size_t &n, const uint32_t *input,
                                                    uint32_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<uint8_t>(const size_t &m, const size_t &n, const uint8_t *input,
                                                   uint8_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<uint64_t>(const size_t &m, const size_t &n, const uint64_t *input,
                                                    uint64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t Fill<bool>(const size_t &m, const size_t &n, const bool *input, bool *output,
                                                hipStream_t stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/soft_shrink_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SoftShrinkComp(size_t size, const T *input, const float lambd, T *output) {
  const T positive_lambd = static_cast<T>(lambd);
  const T negative_lambd = static_cast<T>(-1 * lambd);
  const T zero = static_cast<T>(0);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output[pos] = (input[pos] > positive_lambd)
                    ? (input[pos] - positive_lambd)
                    : ((input[pos] < negative_lambd) ? (input[pos] + positive_lambd) : (zero));
  }
}

template <typename T>
__global__ void SoftShrinkGradComp(size_t size, const T *dy_addr, const T *x_addr, const float lambd, T *dx_addr) {
  const T positive_lambd = static_cast<T>(lambd);
  const T negative_lambd = static_cast<T>(-1 * lambd);
  const T zero = static_cast<T>(0);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    dx_addr[pos] = (x_addr[pos] >= negative_lambd && x_addr[pos] <= positive_lambd) ? zero : dy_addr[pos];
  }
}

template <typename T>
hipError_t SoftShrink(const size_t &size, const T *input, const float lambd, T *output, const uint32_t &device_id,
                       hipStream_t hip_stream) {
  SoftShrinkComp<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input, lambd, output);
  return GetCudaStatus();
}

template <typename T>
hipError_t SoftShrinkGrad(const size_t &size, const T *dy_addr, const T *x_addr, const float lambd, T *dx_addr,
                           const uint32_t &device_id, hipStream_t hip_stream) {
  SoftShrinkGradComp<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, dy_addr, x_addr,
                                                                                                lambd, dx_addr);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t SoftShrink(const size_t &size, const half *input, const float lambd, half *output,
                                                const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrink(const size_t &size, const float *input, const float lambd,
                                                float *output, const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrink(const size_t &size, const int *input, const float lambd, int *output,
                                                const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrink(const size_t &size, const int64_t *input, const float lambd,
                                                int64_t *output, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t SoftShrinkGrad(const size_t &size, const half *dy_addr, const half *x_addr,
                                                    const float lambd, half *dx_addr, const uint32_t &device_id,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrinkGrad(const size_t &size, const float *dy_addr, const float *x_addr,
                                                    const float lambd, float *dx_addr, const uint32_t &device_id,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrinkGrad(const size_t &size, const int *dy_addr, const int *x_addr,
                                                    const float lambd, int *dx_addr, const uint32_t &device_id,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT hipError_t SoftShrinkGrad(const size_t &size, const int64_t *dy_addr, const int64_t *x_addr,
                                                    const float lambd, int64_t *dx_addr, const uint32_t &device_id,
                                                    hipStream_t hip_stream);

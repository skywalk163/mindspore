#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/sparse_matrix_transpose_impl.cuh"
#include <stdint.h>
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

__global__ void ConjKernel(const size_t input_size, hipComplex *input_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < input_size; pos += blockDim.x * gridDim.x) {
    hipComplex *tar = input_addr + pos;
    *tar = hipConjf(*tar);
  }
}

__global__ void ConjKernel(const size_t input_size, hipDoubleComplex *input_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < input_size; pos += blockDim.x * gridDim.x) {
    hipDoubleComplex *tar = input_addr + pos;
    *tar = hipConj(*tar);
  }
}

hipError_t Conj(const size_t input_size, hipComplex *input_addr, hipStream_t stream) {
  ConjKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, input_addr);
  return GetCudaStatus();
}

hipError_t Conj(const size_t input_size, hipDoubleComplex *input_addr, hipStream_t stream) {
  ConjKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, input_addr);
  return GetCudaStatus();
}

CUDA_LIB_EXPORT hipError_t Conj(const size_t input_size, hipComplex *input_addr, hipStream_t stream);
CUDA_LIB_EXPORT hipError_t Conj(const size_t input_size, hipDoubleComplex *input_addr, hipStream_t stream);

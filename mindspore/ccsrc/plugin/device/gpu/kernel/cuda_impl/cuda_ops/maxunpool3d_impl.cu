#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/maxunpool3d_impl.cuh"
template <typename T>
__global__ void InitMaxUnpool3D(const int64_t outer_size, T *output) {
  T zero = 0;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < outer_size; pos += blockDim.x * gridDim.x) {
    output[pos] = zero;
  }
  return;
}
template <typename T, typename S>
__global__ void MaxUnpool3DNCDHW(const T *input, const S *indices, const int64_t inputChannel, const int64_t inputFlame,
                                 const int64_t inputHeight, const int64_t inputWidth, const int64_t outputChannel,
                                 const int64_t outputFlame, const int64_t outputHeight, const int64_t outputWidth,
                                 const int64_t thread_size, T *output) {
  int posn = blockIdx.z;
  int posc = blockIdx.y;
  output += (posn * inputChannel + posc) * outputFlame * outputHeight * outputWidth;
  input += (posn * inputChannel + posc) * inputFlame * inputHeight * inputWidth;
  indices += (posn * inputChannel + posc) * inputFlame * inputHeight * inputWidth;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < thread_size; pos += blockDim.x * gridDim.x) {
    S maxind = indices[pos];
    CUDA_KERNEL_ASSERT(maxind >= 0 && maxind < outputChannel * outputFlame * outputHeight * outputWidth);
    output[maxind] = input[pos];
  }

  return;
}

template <typename T, typename S>
__global__ void MaxUnpool3DNDHWC(const T *input, const S *indices, const int64_t inputFlame, const int64_t inputHeight,
                                 const int64_t inputWidth, const int64_t inputChannel, const int64_t outputFlame,
                                 const int64_t outputHeight, const int64_t outputWidth, const int64_t outputChannel,
                                 const int64_t thread_size, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < thread_size; pos += blockDim.x * gridDim.x) {
    const int posn = pos / (inputHeight * inputWidth * inputFlame * inputChannel);
    const int posc = pos % inputChannel;
    S maxind = indices[pos];
    CUDA_KERNEL_ASSERT(maxind >= 0 && maxind < outputFlame * outputHeight * outputWidth * outputChannel);
    output[(posn * outputFlame * outputHeight * outputWidth + maxind) * outputChannel + posc] = input[pos];
  }

  return;
}

template <typename T, typename S>
hipError_t CalMaxUnpool3D(const T *input, const S *indices, const std::vector<int64_t> input_shape,
                           const std::vector<int64_t> output_shape, T *output, const int64_t outer_size,
                           const int64_t thread_size, const std::string data_format_, const uint32_t &device_id,
                           hipStream_t hip_stream) {
  InitMaxUnpool3D<<<CUDA_BLOCKS(device_id, outer_size), CUDA_THREADS(device_id), 0, hip_stream>>>(outer_size, output);
  if (data_format_ == "NCDHW") {
    int outputPlaneSize = input_shape[2] * input_shape[3] * input_shape[4];
    dim3 grid((outputPlaneSize + 127) / 128, input_shape[1], input_shape[0]);
    dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);
    MaxUnpool3DNCDHW<<<grid, block, 0, hip_stream>>>(input, indices, input_shape[1], input_shape[2], input_shape[3],
                                                      input_shape[4], output_shape[1], output_shape[2], output_shape[3],
                                                      output_shape[4], outputPlaneSize, output);
  } else {
    MaxUnpool3DNDHWC<<<CUDA_BLOCKS(device_id, thread_size), CUDA_THREADS(device_id), 0, hip_stream>>>(
      input, indices, input_shape[1], input_shape[2], input_shape[3], input_shape[4], output_shape[1], output_shape[2],
      output_shape[3], output_shape[4], thread_size, output);
  }
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint8_t, int32_t>(
  const uint8_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint8_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint8_t, int64_t>(
  const uint8_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint8_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint16_t, int32_t>(
  const uint16_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint16_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint16_t, int64_t>(
  const uint16_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint16_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint32_t, int32_t>(
  const uint32_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint32_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint32_t, int64_t>(
  const uint32_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint32_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint64_t, int32_t>(
  const uint64_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint64_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<uint64_t, int64_t>(
  const uint64_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, uint64_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int8_t, int32_t>(
  const int8_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int8_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int8_t, int64_t>(
  const int8_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int8_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int16_t, int32_t>(
  const int16_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int16_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int16_t, int64_t>(
  const int16_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int16_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int32_t, int32_t>(
  const int32_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int32_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int32_t, int64_t>(
  const int32_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int32_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int64_t, int32_t>(
  const int64_t *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int64_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<int64_t, int64_t>(
  const int64_t *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, int64_t *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<half, int32_t>(
  const half *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, half *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<half, int64_t>(
  const half *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, half *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<float, int32_t>(
  const float *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, float *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<float, int64_t>(
  const float *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, float *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<double, int32_t>(
  const double *input, const int32_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, double *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalMaxUnpool3D<double, int64_t>(
  const double *input, const int64_t *indices, const std::vector<int64_t> input_shape,
  const std::vector<int64_t> output_shape, double *output, const int64_t outer_size, const int64_t thread_size,
  const std::string data_format_, const uint32_t &device_id, hipStream_t hip_stream);

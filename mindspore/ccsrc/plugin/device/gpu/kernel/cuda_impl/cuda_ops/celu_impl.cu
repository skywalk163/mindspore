#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/celu_impl.cuh"
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void CalculateCeluKernel(const T *input, const size_t input_elements, double alpha, T *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < input_elements; i += blockDim.x * gridDim.x) {
    T input_value = input[i];
    double inv_alpha = static_cast<double>(1.0) / alpha;
    output[i] = input_value > 0 ? input_value : alpha * std::expm1(input_value * inv_alpha);
  }
}

__global__ void CalculateCeluKernel(const half *input, const size_t input_elements, double alpha, half *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < input_elements; i += blockDim.x * gridDim.x) {
    half input_value = input[i];
    double inv_alpha = static_cast<double>(1.0) / alpha;
    output[i] = input_value > static_cast<half>(0) ? input_value
                                                   : __float2half(alpha * expm1(__half2float(input_value) * inv_alpha));
  }
}

template <typename T>
hipError_t CalculateCelu(const T *input, size_t input_elements, double alpha, T *output, const uint32_t &device_id,
                          hipStream_t hip_stream) {
  CalculateCeluKernel<<<CUDA_BLOCKS(device_id, input_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(
    input, input_elements, alpha, output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalculateCelu<double>(const double *input, size_t input_elements, double alpha,
                                                           double *output, const uint32_t &device_id,
                                                           hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalculateCelu<float>(const float *input, size_t input_elements, double alpha,
                                                          float *output, const uint32_t &device_id,
                                                          hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalculateCelu<half>(const half *input, size_t input_elements, double alpha,
                                                         half *output, const uint32_t &device_id,
                                                         hipStream_t hip_stream);

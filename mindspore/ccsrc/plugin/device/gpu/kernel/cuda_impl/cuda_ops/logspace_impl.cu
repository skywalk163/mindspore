#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/logspace_impl.cuh"
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void LogSpaceKernel(const T *start, const T *end, const int64_t steps, const size_t base, T *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < steps; i += gridDim.x * blockDim.x) {
    T add = (end[0] - start[0]) / (static_cast<T>(steps == 1 ? steps : steps - 1));
    output[i] = pow(static_cast<T>(base), start[0] + (add * i));
  }
  return;
}

template <>
__global__ void LogSpaceKernel(const half *start, const half *end, const int64_t steps, const size_t base,
                               half *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < steps; i += gridDim.x * blockDim.x) {
    float start_float = __half2float(start[0]);
    float add = (__half2float(end[0]) - start_float) / (static_cast<float>(steps == 1 ? steps : steps - 1));
    output[i] = __float2half(pow(static_cast<float>(base), start_float + (add * i)));
  }
  return;
}

template <typename T>
hipError_t CalLogSpace(const T *start, const T *end, const int64_t steps, const size_t base, T *output,
                        const uint32_t &device_id, hipStream_t hip_stream) {
  LogSpaceKernel<<<CUDA_BLOCKS(device_id, steps), CUDA_THREADS(device_id), 0, hip_stream>>>(start, end, steps, base,
                                                                                             output);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t CalLogSpace<half>(const half *start, const half *end, const int64_t steps,
                                                       const size_t base, half *output, const uint32_t &device_id,
                                                       hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalLogSpace<float>(const float *start, const float *end, const int64_t steps,
                                                        const size_t base, float *output, const uint32_t &device_id,
                                                        hipStream_t hip_stream);

template CUDA_LIB_EXPORT hipError_t CalLogSpace<double>(const double *start, const double *end, const int64_t steps,
                                                         const size_t base, double *output, const uint32_t &device_id,
                                                         hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/normalize_impl.cuh"

#define FINAL_MASK 0xffffffff

template <typename T>
__device__ T warpedReduceSum(T val) {
#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  }
  return val;
}

template <typename T>
__device__ T blockReduceSum(T val) {
  static __shared__ T shared[32];
  int warped = threadIdx.x & 0x1f;
  val = warpedReduceSum<T>(val);
  if (warped == 0) shared[threadIdx.x >> 5] = val;
  __syncthreads();
  val = (threadIdx.x < (blockDim.x / 32.f)) ? shared[warped] : static_cast<T>(0.0);
  val = warpedReduceSum<T>(val);
  return val;
}

template <typename T>
__global__ void NormalizeKernel(const T *input, const T *gamma, const T *beta, T *output, size_t n, float epsilion,
                                int dim_before_axis) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int block_loop = (dim_before_axis - 1) / gridDim.x + 1;
  const int element_cnt = dim_before_axis * n;

  __shared__ float s_mean[2048];
  __shared__ float s_variance[2048];
  float sum = 0.0f;
  float variance = 0.0f;

  for (int block = 0; block < block_loop; block++) {
    float local_sum = 0.0f;
    int mean_index = bid + block * gridDim.x;
    int num_index = bid * n + block * gridDim.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x) {
      if (num_index + i >= element_cnt) {
        break;
      }
      local_sum += static_cast<float>(input[num_index + i]);
    }
    sum = blockReduceSum(local_sum);
    if (tid == 0) {
      s_mean[mean_index] = sum / n;
    }
  }
  __syncthreads();

  for (int block = 0; block < block_loop; block++) {
    float local_var_sum = 0.0f;
    int var_index = bid + block * gridDim.x;
    int num_index = bid * n + block * gridDim.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x) {
      if (num_index + i >= element_cnt) {
        break;
      }
      float diff = static_cast<float>(input[num_index + i]) - s_mean[var_index];
      local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);
    if (tid == 0) {
      s_variance[var_index] = rsqrtf(variance / n + epsilion);
    }
  }
  __syncthreads();
  for (int block = 0; block < block_loop; block++) {
    int var_index = bid + block * gridDim.x;
    int num_index = bid * n + block * gridDim.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x) {
      if (num_index + i >= element_cnt) {
        break;
      }
      float beta_val = (beta == nullptr) ? 0.0f : static_cast<float>(beta[i]);
      output[num_index + i] =
        static_cast<T>(((static_cast<float>(input[num_index + i]) - s_mean[var_index]) * s_variance[var_index]) *
                         static_cast<float>(gamma[i]) +
                       beta_val);
    }
  }
}

template <typename T>
hipError_t Normalize(const T *input, const T *gamma, const T *beta, T *output, size_t dim_at_axis, float epsilion,
                      int element_cnt, hipStream_t stream, const uint32_t device_id) {
  int threads_num = CUDA_THREADS_MAXSIZE(device_id, ((dim_at_axis - 1) / 32 + 1) * 32);
  int blocks_num = CUDA_BLOCKS_CAL(device_id, element_cnt, threads_num);
  int dim_before_axis = element_cnt / dim_at_axis;
  NormalizeKernel<<<blocks_num, threads_num, 0, stream>>>(input, gamma, beta, output, dim_at_axis, epsilion,
                                                          dim_before_axis);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t Normalize(const float *input, const float *gamma, const float *beta, float *output,
                                               size_t dim_at_axis, float epsilion, int element_cnt, hipStream_t stream,
                                               const uint32_t device_id);

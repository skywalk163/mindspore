#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/rl/priority_replay_buffer.cuh"
#include <hip/hip_runtime_api.h>
#include <thrust/detail/minmax.h>
#include <limits>
#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

constexpr float kInitPriority = 1.0;
constexpr size_t kRootIdx = 1;
constexpr size_t kNumSubNode = 2;
constexpr size_t kMaxThreadPerBlock = 128;

__global__ void InitRandStateKernel(uint64_t seed, hiprandState *state) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &(state[tid]));
}

hipError_t InitRandState(const size_t &batch_size, const uint64_t &seed, hiprandState *state, hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  InitRandStateKernel<<<grid, block, 0, stream>>>(seed, state);
  return GetCudaStatus();
}

__global__ void FifoSliceKernel(const uint8_t *input, const size_t *indice, uint8_t *output, size_t batch_size,
                                size_t column) {
  for (size_t num = blockIdx.x * blockDim.x + threadIdx.x; num < batch_size * column; num += blockDim.x * gridDim.x) {
    size_t i = num / column;
    size_t j = num % column;
    size_t read_idex = indice[i] * column + j;
    output[num] = input[read_idex];
  }
}

hipError_t FifoSlice(const uint8_t *input, const size_t *indice, uint8_t *output, size_t batch_size, size_t column,
                      hipStream_t stream) {
  size_t num = batch_size * column;
  size_t block = std::min(num, kMaxThreadPerBlock);
  size_t grid = (num + block - 1) / block;
  FifoSliceKernel<<<grid, block, 0, stream>>>(input, indice, output, batch_size, column);
  return GetCudaStatus();
}

template <typename T>
__global__ void SumTreeInitKernel(T *tree, float *max_priority, size_t size) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    tree[i].init();

    if (i == 0) {
      *max_priority = kInitPriority;
    }
  }
}

template <typename T>
__forceinline__ __device__ void SumTreeInsert(T *tree, size_t idx, float priority) {
  tree[idx].set(priority);

  size_t parent = idx >> 1;
  while (parent >= kRootIdx) {
    size_t left_child = parent << 1;
    tree->reduce(tree + left_child, tree + left_child + 1, tree + parent);
    parent >>= 1;
  }
}

template <typename T>
__global__ void SumTreePushKernel(T *tree, float alpha, size_t idx, float *priority, float *max_priority) {
  float prio;
  if (!priority) {
    prio = powf(*max_priority, alpha);
  } else {
    *max_priority = thrust::max(*max_priority, *priority);
    prio = powf(*priority, alpha);
  }

  SumTreeInsert(tree, idx, prio);
}

template <typename T>
__forceinline__ __device__ size_t GetPrefixSumIdx(T *tree, size_t capacity, float prefix_sum) {
  size_t idx = kRootIdx;
  while (idx < capacity) {
    const float &left_priority = tree[kNumSubNode * idx].sum;
    if (prefix_sum <= left_priority) {
      idx = kNumSubNode * idx;
    } else {
      prefix_sum -= left_priority;
      idx = kNumSubNode * idx + 1;
    }
  }
  return idx - capacity;
}

template <typename T>
__global__ void SumTreeSampleKernel(T *tree, hiprandState *state, size_t capacity, float *beta, size_t batch_size,
                                    size_t *indices, float *weights) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += gridDim.x * blockDim.x) {
    size_t segment_len = tree[kRootIdx].sum / batch_size;
    float prefix_sum = (hiprand_uniform(&state[i]) + i) * segment_len;
    size_t idx = GetPrefixSumIdx(tree, capacity, prefix_sum);
    indices[i] = idx;
    weights[i] = powf((tree[idx + capacity].sum / tree[kRootIdx].min), -beta[0]);
  }
}

__global__ void SumTreeGetGlobalIdxKernel(size_t batch_size, size_t *indices, size_t total_num, size_t capacity) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += gridDim.x * blockDim.x) {
    size_t idx = indices[i] + (total_num - total_num % capacity);
    if (idx > total_num) {
      idx -= capacity;
    }
    indices[i] = idx;
  }
}

template <typename T>
__global__ void SumTreeUpdateKernel(T *tree, size_t capacity, size_t last_idx, float alpha, float *max_priority,
                                    size_t *indices, float *priorities, size_t batch_size) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += gridDim.x * blockDim.x) {
    size_t idx = indices[i];
    // skip if the transition is already replaced.
    if (idx < last_idx) continue;

    float priority = powf(priorities[i], alpha);
    MsAtomicMax(max_priority, priority);

    idx += -last_idx + capacity;
    SumTreeInsert(tree, idx, priority);
  }
}

// Init Segment Tree: Fill all item with the specified value
template <typename T>
hipError_t SumTreeInit(T *tree, float *max_priority, const size_t &capacity, hipStream_t stream) {
  size_t size = capacity * kNumSubNode;
  size_t block = std::min(size, kMaxThreadPerBlock);
  size_t grid = (size + block - 1) / block;
  SumTreeInitKernel<<<grid, block, 0, stream>>>(tree, max_priority, size);
  return GetCudaStatus();
}

// Push one item to the tree. Set item with max_priority if the priority not provided.
template <typename T>
hipError_t SumTreePush(T *tree, const float &alpha, const size_t &idx, const size_t &capacity, float *priority,
                        float *max_priority, hipStream_t stream) {
  size_t idx_in_tree = idx + capacity;
  SumTreePushKernel<<<1, 1, 0, stream>>>(tree, alpha, idx_in_tree, priority, max_priority);
  return GetCudaStatus();
}

// Sample a batch item. Return indices and correction weights.
template <typename T>
hipError_t SumTreeSample(T *tree, hiprandState *state, const size_t &capacity, float *beta, const size_t &batch_size,
                          size_t *indices, float *weights, hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  SumTreeSampleKernel<<<grid, block, 0, stream>>>(tree, state, capacity, beta, batch_size, indices, weights);
  return GetCudaStatus();
}

hipError_t SumTreeGetGlobalIdx(size_t batch_size, size_t *indices, size_t total_num, size_t capacity,
                                hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  SumTreeGetGlobalIdxKernel<<<grid, block, 0, stream>>>(batch_size, indices, total_num, capacity);
  return GetCudaStatus();
}

// Update item priority.
template <typename T>
hipError_t SumTreeUpdate(T *tree, const size_t &capacity, const size_t &last_idx, const float &alpha,
                          float *max_priority, size_t *indices, float *priorities, const size_t &batch_size,
                          hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  SumTreeUpdateKernel<<<grid, block, 0, stream>>>(tree, capacity, last_idx, alpha, max_priority, indices, priorities,
                                                  batch_size);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t SumTreeInit<SumMinTree>(SumMinTree *tree, float *max_priority,
                                                             const size_t &capacity, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t SumTreePush<SumMinTree>(SumMinTree *tree, const float &alpha, const size_t &idx,
                                                             const size_t &capacity, float *priority,
                                                             float *max_priority, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t SumTreeSample<SumMinTree>(SumMinTree *tree, hiprandState *state,
                                                               const size_t &capacity, float *beta,
                                                               const size_t &batch_size, size_t *indices,
                                                               float *weights, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t SumTreeUpdate<SumMinTree>(SumMinTree *tree, const size_t &capacity,
                                                               const size_t &last_idx, const float &alpha,
                                                               float *max_priority, size_t *indices, float *priorities,
                                                               const size_t &batch_size, hipStream_t stream);
